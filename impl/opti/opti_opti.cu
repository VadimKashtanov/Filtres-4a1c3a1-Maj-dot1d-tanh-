#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static uint * tout_zeroiser = UNIFORME_C(1);

PAS_OPTIMISER()
void __interne_optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	uint ** masque,
	uint ** masque_opti,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C],
	uint _t_MODE, uint GRAINE)
{
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	//
	//	Cree les listes pour les `hist` si un opti en a besoin 
	Opti_classe_t opti_classe;
	if      (methode == SGD)     opti_classe.sgd     = (uint)NULL;
	else if (methode == RMSPROP) opti_classe.rmsprop = cree_rmsprop(mdl);
	else if (methode == ADAM)    opti_classe.adam    = cree_adam(mdl);
	else ERR("Pas de methode %i d'optimisation", methode);
	
	//	Plumer grad pour mieux y voire
	mdl_plume_grad(mdl, t0, t1, _t_MODE, GRAINE);
	
	/* ------- Optimisation ----------- */
	uint zeroiser[C];
	FOR(0, i, I) {
		//
		FOR(0, j, C) {
			if (i % zero_accumulation_tous_les[j] == 0)
				zeroiser[j] = 1;
			else
				zeroiser[j] = 0;
		}
		//
		perturber(mdl, PERTURBATIONS);
		mdl_aller_retour(mdl, t0, t1, 3, _t_MODE, GRAINE);
		
		//	--------- * Optimisation * -------------
#define optimiser_la_couche zeroiser
		if (methode == SGD)     opti_simple (zero_accumulation_tous_les, optimiser_la_couche, mdl, alpha, div, masque, masque_opti);
		if (methode == RMSPROP) opti_rmsprop(zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.rmsprop, alpha, div, masque, masque_opti);
		if (methode == ADAM)    opti_adam   (zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.adam,    alpha, div, masque, masque_opti);
		//
		mdl_zero_deriv_gpu(mdl, zeroiser);
		//
		mdl_normer_les_filtres(mdl);
		//
		if (i % /*5*/100 == 0) {
			//mdl_plume_grad(mdl, t0, t1, _t_MODE, GRAINE);
			//
			float* __pred = mdl_pred(mdl, t0, t1, 3, _t_MODE, GRAINE);
			float  _score = mdl_score(mdl, t0, t1, 3, _t_MODE, GRAINE);
			//
			float les_gains__2 = mdl_les_gains(mdl, t0, t1, 3,   2.0, _t_MODE, GRAINE);
			float les_gains__4 = mdl_les_gains(mdl, t0, t1, 3,   4.0, _t_MODE, GRAINE);
			//
			printf("%3.i/%3.i| perf={", i, I);
			FOR(0, p, P) printf("%+f%%, ", 100*__pred[p]);
			free(__pred);
			printf("} score=\033[93m%+f\033[0m (%%.potentiel^2=%+f, %%.potentiel^4=%+f)\n",
				_score,
				les_gains__2,
				les_gains__4
			);
			if (fabs(_score) < 0.00001) {
				printf("Score < 0.00001 => Fin d'optimisation\n");
				break;
			}
		}
	}

	//	Liberer
	if      (methode == SGD)     opti_classe.sgd = 0;
	else if (methode == RMSPROP) liberer_rmsprop(opti_classe.rmsprop);
	else if (methode == ADAM)    liberer_adam   (opti_classe.adam   );
};

void optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	float * pourcent_masque,
	float * pourcent_opti_masque,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C],
	uint _t_MODE, uint GRAINE)
{
	Masque_t * masque = cree_masque(mdl, pourcent_masque, pourcent_opti_masque);
	//
	__interne_optimiser(
		mdl,
		t0, t1,
		alpha, div,
		methode, I,
		masque->masque,
		masque->masque_opti,
		PERTURBATIONS,
		zero_accumulation_tous_les,
		_t_MODE, GRAINE);
	//
	sortire_masque(mdl, masque);
};