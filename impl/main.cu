#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

uint MODE_t_MODE = t_CONTINUE;
uint grain_t_MODE = 0;

static void visualiser() {
	uint source     = SRC_PRIXS;
	uint nature     = POURCENT_R;
	uint K_ema      = 64;
	uint intervalle = 256;
	uint * params   = cree_POURCENT_R(256, 2);
	visualiser_ema_int(
		source,
		nature,
		K_ema, intervalle,
		params);
};

static void plume_pred(Mdl_t * mdl, uint t0, uint t1) {
	MODE_t_MODE = t_CONTINUE;
	grain_t_MODE = 0;
	//
	uint fois = (t1-t0)/mdl->T;
	//
	float moyenne_pred[P] = {0};
	float moyenne_les_gain__2 = 0; float coef__2 = 2.0;
	float moyenne_les_gain__4 = 0; float coef__4 = 4.0;
	float moyenne_les_gain__8 = 0; float coef__8 = 8.0;
	//
	FOR(0, i, fois) {
		float * ancien = mdl_pred(mdl, t0 + i*mdl->T, t0 + (i+1)*mdl->T, 3, MODE_t_MODE, grain_t_MODE);
		FOR(0, p, P) moyenne_pred[p] += ancien[p];
		free(ancien);
		//
		moyenne_les_gain__2 += mdl_les_gains(mdl, t0 + i*mdl->T, t0 + (i+1)*mdl->T, 3, coef__2, MODE_t_MODE, grain_t_MODE);
		moyenne_les_gain__4 += mdl_les_gains(mdl, t0 + i*mdl->T, t0 + (i+1)*mdl->T, 3, coef__4, MODE_t_MODE, grain_t_MODE);
		moyenne_les_gain__8 += mdl_les_gains(mdl, t0 + i*mdl->T, t0 + (i+1)*mdl->T, 3, coef__8, MODE_t_MODE, grain_t_MODE);
	}
	printf("PRED GENERALE = ");
	FOR(0, p, P) printf(" %f%% ", 100*moyenne_pred[p]/(float)fois);
	printf("  | LES GAINS^2 = %f%% | LES GAINS^4 = %f%% | LES GAINS^8 = %f%%",
		100*moyenne_les_gain__2/(float)fois,
		100*moyenne_les_gain__4/(float)fois,
		100*moyenne_les_gain__8/(float)fois
	);
	printf("\n");
};

float pourcent_masque_nulle[C] = {0};
float pourcent_masque_opti_nulle[C] = {0};

float * pourcent_masque = de_a(0.40, 0.10, C);			//	Des poids nulls
float * pourcent_masque_opti = de_a(0.10, 0.05, C);		//	Des poids qui ne s'optimiseront pas

float * alpha = de_a(1e-5, 1e-5, C);

uint * optimiser_tous_les = UNIFORME_C(1);

#define GRAND_T (16*16*7)

//	-------------- logistique, tanh et exp(-x**2)

#define EMAISATION 10

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	alpha[0] = 1e-5;

	//pourcent_masque[0] = 0.70;
	//pourcent_masque[1] = 0.01;

	//	----- Lien constants ------

	//	Faire des instructions cascades et de regard de reflexion (les couches)
	//	sans passer le gradient, donc x constant

	//pourcent_masque_nulle[0] = 0.30;
	//pourcent_masque_nulle[1] = 0.70;

	/*
		* VAPP
	*/
	
	//	-- Init --
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");   charger_tout();

	//	-- Verification --
	titre("Verifier MDL");     verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");
	ecrire_structure_generale("structure_generale.bin");

	//visualiser();

	uint Y[C];
	uint insts[C];
	uint cible[C];
	uint decale_future[C];
	//
	uint st[C][4] = {
		//{4096, DOT1D,          CIBLE_FILTRES_FUTURES,   1 },
		//
	//	  Y      inst             cible             decale-future
		{2048, FILTRES_PRIXS_TOLERANT, 	CIBLE_NORMALE, NULL},
		{16,   DOT1D_TANH,             	CIBLE_NORMALE, NULL},
		//
		{64,   DOT1D_LOGISTIC,			CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		{64,   DOT1D_LOGISTIC,          CIBLE_NORMALE, NULL},
		//
		{P,    DOT1D_TANH,              CIBLE_NORMALE, NULL}
	};
	FOR(0, i, C) {
		    Y[i] = st[i][0];
		insts[i] = st[i][1];
		cible[i] = st[i][2];
		decale_future[i] = st[i][3];
	}
	//
	//	Assurances :
	ema_int_t * bloque[BLOQUES] = {
	//			    Source,      Nature,  K_ema, Intervalle,     {params}
		cree_ligne(SRC_PRIXS, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 1, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 2, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 4, 16, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 8, 32, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 16, 64, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 32, 128, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 64, 256, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_PRIXS, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 1, 4, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 2, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 4, 16, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 8, 32, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 16, 64, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 32, 128, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 64, 256, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_HIGH, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 1, 4, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 2, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 4, 16, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 8, 32, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 16, 64, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 32, 128, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 64, 256, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_LOW, DIRECT, 256, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 1, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 1, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 2, 1.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 2, 2, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 2, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 2, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 2.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 4, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 4, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 4.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 8, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 8, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 8.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 16, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 16, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 32, 16.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 32, 32, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 32, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 32, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 32.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 64, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 64, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 64.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 128, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 128, 256, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 128.0, cree_DIRECTE()),
		cree_ligne(SRC_VOLUMES, DIRECT, 256, 256, cree_DIRECTE()),
	// -------
		cree_ligne(SRC_PRIXS, MACD, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 1, 8, cree_MACD(4)),
		cree_ligne(SRC_PRIXS, MACD, 2, 2, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 2, 16, cree_MACD(2)),
		cree_ligne(SRC_PRIXS, MACD, 2, 16, cree_MACD(8)),
		cree_ligne(SRC_PRIXS, MACD, 4, 2.0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 4, 4, cree_MACD(2)),
		cree_ligne(SRC_PRIXS, MACD, 4, 32, cree_MACD(4)),
		cree_ligne(SRC_PRIXS, MACD, 4, 32, cree_MACD(16)),
		cree_ligne(SRC_PRIXS, MACD, 8, 4.0, cree_MACD(2)),
		cree_ligne(SRC_PRIXS, MACD, 8, 8, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 8, 64, cree_MACD(8)),
		cree_ligne(SRC_PRIXS, MACD, 8, 64, cree_MACD(32)),
		cree_ligne(SRC_PRIXS, MACD, 16, 8.0, cree_MACD(1)),
		cree_ligne(SRC_PRIXS, MACD, 16, 8.0, cree_MACD(4)),
		cree_ligne(SRC_PRIXS, MACD, 16, 16, cree_MACD(8)),
		cree_ligne(SRC_PRIXS, MACD, 16, 128, cree_MACD(16)),
		cree_ligne(SRC_PRIXS, MACD, 16, 128, cree_MACD(64)),
		cree_ligne(SRC_PRIXS, MACD, 32, 16.0, cree_MACD(2)),
		cree_ligne(SRC_PRIXS, MACD, 32, 16.0, cree_MACD(8)),
		cree_ligne(SRC_PRIXS, MACD, 32, 32, cree_MACD(4)),
		cree_ligne(SRC_PRIXS, MACD, 32, 32, cree_MACD(16)),
		cree_ligne(SRC_PRIXS, MACD, 32, 256, cree_MACD(128)),
		cree_ligne(SRC_PRIXS, MACD, 64, 32.0, cree_MACD(4)),
		cree_ligne(SRC_PRIXS, MACD, 64, 32.0, cree_MACD(16)),
		cree_ligne(SRC_PRIXS, MACD, 64, 64, cree_MACD(8)),
		cree_ligne(SRC_PRIXS, MACD, 256, 128.0, cree_MACD(16)),
		cree_ligne(SRC_PRIXS, MACD, 256, 128.0, cree_MACD(64)),
		cree_ligne(SRC_PRIXS, MACD, 256, 256, cree_MACD(32)),
		cree_ligne(SRC_PRIXS, MACD, 256, 256, cree_MACD(128)),
		cree_ligne(SRC_HIGH, MACD, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 1, 8, cree_MACD(4)),
		cree_ligne(SRC_HIGH, MACD, 2, 2, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 2, 16, cree_MACD(2)),
		cree_ligne(SRC_HIGH, MACD, 2, 16, cree_MACD(8)),
		cree_ligne(SRC_HIGH, MACD, 4, 2.0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 4, 4, cree_MACD(2)),
		cree_ligne(SRC_HIGH, MACD, 4, 32, cree_MACD(4)),
		cree_ligne(SRC_HIGH, MACD, 4, 32, cree_MACD(16)),
		cree_ligne(SRC_HIGH, MACD, 8, 4.0, cree_MACD(2)),
		cree_ligne(SRC_HIGH, MACD, 8, 8, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 8, 64, cree_MACD(8)),
		cree_ligne(SRC_HIGH, MACD, 8, 64, cree_MACD(32)),
		cree_ligne(SRC_HIGH, MACD, 16, 8.0, cree_MACD(1)),
		cree_ligne(SRC_HIGH, MACD, 16, 8.0, cree_MACD(4)),
		cree_ligne(SRC_HIGH, MACD, 16, 16, cree_MACD(8)),
		cree_ligne(SRC_HIGH, MACD, 16, 128, cree_MACD(16)),
		cree_ligne(SRC_HIGH, MACD, 16, 128, cree_MACD(64)),
		cree_ligne(SRC_HIGH, MACD, 32, 16.0, cree_MACD(2)),
		cree_ligne(SRC_HIGH, MACD, 32, 16.0, cree_MACD(8)),
		cree_ligne(SRC_HIGH, MACD, 32, 32, cree_MACD(4)),
		cree_ligne(SRC_HIGH, MACD, 32, 32, cree_MACD(16)),
		cree_ligne(SRC_HIGH, MACD, 32, 256, cree_MACD(128)),
		cree_ligne(SRC_HIGH, MACD, 64, 32.0, cree_MACD(4)),
		cree_ligne(SRC_HIGH, MACD, 64, 32.0, cree_MACD(16)),
		cree_ligne(SRC_HIGH, MACD, 64, 64, cree_MACD(8)),
		cree_ligne(SRC_HIGH, MACD, 256, 128.0, cree_MACD(16)),
		cree_ligne(SRC_HIGH, MACD, 256, 128.0, cree_MACD(64)),
		cree_ligne(SRC_HIGH, MACD, 256, 256, cree_MACD(32)),
		cree_ligne(SRC_HIGH, MACD, 256, 256, cree_MACD(128)),
		cree_ligne(SRC_LOW, MACD, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 1, 8, cree_MACD(4)),
		cree_ligne(SRC_LOW, MACD, 2, 2, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 2, 16, cree_MACD(2)),
		cree_ligne(SRC_LOW, MACD, 2, 16, cree_MACD(8)),
		cree_ligne(SRC_LOW, MACD, 4, 2.0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 4, 4, cree_MACD(2)),
		cree_ligne(SRC_LOW, MACD, 4, 32, cree_MACD(4)),
		cree_ligne(SRC_LOW, MACD, 4, 32, cree_MACD(16)),
		cree_ligne(SRC_LOW, MACD, 8, 4.0, cree_MACD(2)),
		cree_ligne(SRC_LOW, MACD, 8, 8, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 8, 64, cree_MACD(8)),
		cree_ligne(SRC_LOW, MACD, 8, 64, cree_MACD(32)),
		cree_ligne(SRC_LOW, MACD, 16, 8.0, cree_MACD(1)),
		cree_ligne(SRC_LOW, MACD, 16, 8.0, cree_MACD(4)),
		cree_ligne(SRC_LOW, MACD, 16, 16, cree_MACD(8)),
		cree_ligne(SRC_LOW, MACD, 16, 128, cree_MACD(16)),
		cree_ligne(SRC_LOW, MACD, 16, 128, cree_MACD(64)),
		cree_ligne(SRC_LOW, MACD, 32, 16.0, cree_MACD(2)),
		cree_ligne(SRC_LOW, MACD, 32, 16.0, cree_MACD(8)),
		cree_ligne(SRC_LOW, MACD, 32, 32, cree_MACD(4)),
		cree_ligne(SRC_LOW, MACD, 32, 32, cree_MACD(16)),
		cree_ligne(SRC_LOW, MACD, 32, 256, cree_MACD(128)),
		cree_ligne(SRC_LOW, MACD, 64, 32.0, cree_MACD(4)),
		cree_ligne(SRC_LOW, MACD, 64, 32.0, cree_MACD(16)),
		cree_ligne(SRC_LOW, MACD, 64, 64, cree_MACD(8)),
		cree_ligne(SRC_LOW, MACD, 256, 128.0, cree_MACD(16)),
		cree_ligne(SRC_LOW, MACD, 256, 128.0, cree_MACD(64)),
		cree_ligne(SRC_LOW, MACD, 256, 256, cree_MACD(32)),
		cree_ligne(SRC_LOW, MACD, 256, 256, cree_MACD(128)),
		cree_ligne(SRC_VOLUMES, MACD, 1, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 1, 8, cree_MACD(4)),
		cree_ligne(SRC_VOLUMES, MACD, 2, 2, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 2, 16, cree_MACD(2)),
		cree_ligne(SRC_VOLUMES, MACD, 2, 16, cree_MACD(8)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 2.0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 4, cree_MACD(2)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 32, cree_MACD(4)),
		cree_ligne(SRC_VOLUMES, MACD, 4, 32, cree_MACD(16)),
		cree_ligne(SRC_VOLUMES, MACD, 8, 4.0, cree_MACD(2)),
		cree_ligne(SRC_VOLUMES, MACD, 8, 8, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 8, 64, cree_MACD(8)),
		cree_ligne(SRC_VOLUMES, MACD, 8, 64, cree_MACD(32)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 8.0, cree_MACD(1)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 8.0, cree_MACD(4)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 16, cree_MACD(8)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 128, cree_MACD(16)),
		cree_ligne(SRC_VOLUMES, MACD, 16, 128, cree_MACD(64)),
		cree_ligne(SRC_VOLUMES, MACD, 32, 16.0, cree_MACD(2)),
		cree_ligne(SRC_VOLUMES, MACD, 32, 16.0, cree_MACD(8)),
		cree_ligne(SRC_VOLUMES, MACD, 32, 32, cree_MACD(4)),
		cree_ligne(SRC_VOLUMES, MACD, 32, 32, cree_MACD(16)),
		cree_ligne(SRC_VOLUMES, MACD, 32, 256, cree_MACD(128)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 32.0, cree_MACD(4)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 32.0, cree_MACD(16)),
		cree_ligne(SRC_VOLUMES, MACD, 64, 64, cree_MACD(8)),
		cree_ligne(SRC_VOLUMES, MACD, 256, 128.0, cree_MACD(16)),
		cree_ligne(SRC_VOLUMES, MACD, 256, 128.0, cree_MACD(64)),
		cree_ligne(SRC_VOLUMES, MACD, 256, 256, cree_MACD(32)),
		cree_ligne(SRC_VOLUMES, MACD, 256, 256, cree_MACD(128)),
	};
	//
	Mdl_t * mdl = cree_mdl(GRAND_T, Y, insts, cible, decale_future, bloque);

	//Mdl_t * mdl = ouvrire_mdl(GRAND_T, "mdl.bin");

	//mdl_re_cree_poids(mdl);

	//uint c=5, nouveau_Y=64;
	//mdl_changer_couche_Y(mdl, c, nouveau_Y);

	enregistrer_les_lignes_brute(mdl, "lignes_brute.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = t0 + ROND_MODULO((FIN-DEPART), (16*16));
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%(16*16)=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%(16*16));
	//
	plume_pred(mdl, t0, t1);
	//comportement(mdl, t0, t0+GRAND_T);
	//
	srand(time(NULL));
#define PERTURBATIONS 0
	//
	uint rep = 0;
	while (1) {
		//perturber(mdl, 1000);
		//perturber_filtres(mdl, 10);
		//
		MODE_t_MODE = t_CONTINUE;
		//MODE_t_MODE = t_PSEUDO_ALEA;
		//MODE_t_MODE = t_PSEUDO_ALEA_x16;
		//
		grain_t_MODE = rand() % 10000;
		//
		//
		//if (rand()%10 == 0) alpha[0] = 1e-3;
		//else                alpha[0] = 1e-5;
		//
		//
		optimisation_mini_packet(
			mdl,
			t0, t1, GRAND_T,
			alpha, 1.0,
			ADAM, 1000,//5000,
			//
			//pourcent_masque,
			pourcent_masque_nulle,
			//
			//pourcent_masque_opti,
			pourcent_masque_opti_nulle,
			//
			PERTURBATIONS,
			optimiser_tous_les,
			MODE_t_MODE, grain_t_MODE);
		//
		mdl_poids_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		if (rep % 10 == 0) plume_pred(mdl, t0, t1);
		//
		printf("===================================================\n");
		printf("================= TERMINE %i ======================\n", rep++);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};