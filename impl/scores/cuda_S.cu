#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define pseudo_alea_d_une_grain(i) ((float)((121+(i%1234))*31 % 1001 ) / 1001.0)

//	===============================================================

static __global__ void kerd_nvidia_score_somme(
	uint _t_MODE, uint GRAINE,
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	uint t = threadIdx.x + blockIdx.x + blockDim.x;
	if (t < T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);

		float s = 0;
		FOR(0, p, P) {
			float _y = y[(0 + t)*P + p];
			float alea = 2*pseudo_alea_d_une_grain(t + ((uint)_y % 10001))-1;
			_y += alea * SCORE_Y_COEF_BRUIT;
			s += (P-p)*cuda_SCORE(
				_y, _PRIXS[/*depart+t*/cuda_depart_plus_t+p+1], _PRIXS[/*depart+t*/cuda_depart_plus_t], alea * SCORE_Y_COEF_BRUIT
			);
		}
		atomicAdd(score, s);
	}
};

float nvidia_somme_score(float * y, uint depart, uint T, uint _t_MODE, uint GRAINE)
{
	float * somme_score__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(somme_score__d, 0, sizeof(float)*1));
	kerd_nvidia_score_somme<<<dim3(KERD(T,1)),dim3(1)>>>(
		_t_MODE, GRAINE,
		y, depart, T,
		somme_score__d, prixs__d
	);
	ATTENDRE_CUDA();
	float somme_score;
	CONTROLE_CUDA(hipMemcpy(&somme_score, somme_score__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipFree(somme_score__d));
	return somme_score;
};

float  nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	return APRES_SCORE(somme / (float)(P * T));
};

//	===============================================================

float d_nvidia_score_finale(float somme, uint T, uint _t_MODE, uint GRAINE) {
	return dAPRES_SCORE(somme / (float)(P * T)) / (float)(P * T);
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	uint _t_MODE, uint GRAINE,
	float _dy, float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		FOR(0, p, P) {
			float _y = y[(0+_t)*P+p];
			float alea = 2*pseudo_alea_d_une_grain(_t + ((uint)_y % 10001))-1;
			dy[(0+_t)*P+p] = _dy * (P-p)*cuda_dSCORE(
				y[(0+_t)*P+p]+alea*SCORE_Y_COEF_BRUIT, _PRIXS[/*depart+_t*/cuda_depart_plus_t+p+1], _PRIXS[/*depart+_t*/cuda_depart_plus_t/*+p*/], alea*SCORE_Y_COEF_BRUIT
			);
		}
	}
};

void d_nvidia_somme_score(float d_score, float * y, float * dy, uint depart, uint T, uint _t_MODE, uint GRAINE) {
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		_t_MODE, GRAINE,
		d_score,
		y, dy,
		depart, T,
		prixs__d
	);
	ATTENDRE_CUDA();
};