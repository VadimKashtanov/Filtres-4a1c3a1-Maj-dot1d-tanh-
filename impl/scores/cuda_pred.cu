#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_nvidia_prediction_somme(
	uint _t_MODE, uint GRAINE,
	float * y, uint depart, uint T,
	float * pred, float * _PRIXS,
	uint canal_p)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < T) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, thx);
		float p1 = _PRIXS[/*depart+thx*/cuda_depart_plus_t+canal_p+1];
		float p0 = _PRIXS[/*depart+thx*/cuda_depart_plus_t/*+canal_p*/];
		atomicAdd(
			pred,
			1.0*(uint)(cuda_signe((y[(0+thx)*P+canal_p])) == cuda_signe((p1/p0-1)))
		);
	};
};

static float __nvidia_prediction(float * y, uint depart, uint T, uint canal_p, uint _t_MODE, uint GRAINE) {
	float * pred__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(pred__d, 0, 1*sizeof(float)));
	kerd_nvidia_prediction_somme<<<dim3(KERD(T,1024)),dim3(1024)>>>(
		_t_MODE, GRAINE,
		y, depart, T,
		pred__d, prixs__d,
		canal_p
	);
	ATTENDRE_CUDA();
	float _pred;
	CONTROLE_CUDA(hipMemcpy(&_pred, pred__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	cudafree<float>(pred__d);
	return _pred / (float)T;
};

float * nvidia_prediction(float * y, uint depart, uint T, uint _t_MODE, uint GRAINE) {
	float * pred = (float*)malloc(sizeof(float) * P);
	FOR(0, p, P) pred[p] = __nvidia_prediction(y, depart, T, p, _t_MODE, GRAINE);
	return pred;
};