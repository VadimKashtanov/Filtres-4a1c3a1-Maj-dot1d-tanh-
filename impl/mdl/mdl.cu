#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "filtres_prixs.cuh"
#include "filtres_prixs_tolerant.cuh"
#include "dot1d_tanh.cuh"
#include "dot1d_logistic.cuh"
#include "dot1d_blk.cuh"
#include "dot1d_mul.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

mdl_inst_f cree_inst[INSTS] = {
	cree_filtres_prixs,
	cree_filtres_prixs_tolerant,
	cree_dot1d_tanh,
	cree_dot1d_logistic,
	cree_dot1d_blk,
	cree_dot1d_mul
};

mdl_f_f inst_f [INSTS] = {
	f_filtres_prixs,
	f_filtres_prixs_tolerant,
	f_dot1d_tanh,
	f_dot1d_logistic,
	f_dot1d_blk,
	f_dot1d_mul
};

mdl_f_f inst_df[INSTS] = {
	df_filtres_prixs,
	df_filtres_prixs_tolerant,
	df_dot1d_tanh,
	df_dot1d_logistic,
	df_dot1d_blk,
	df_dot1d_mul
};

char * nom_inst[INSTS] = {
	"filtres_prixs         ",
	"filtres_prixs tolerant",
	"dot1d tanh(x)         ",
	"dot1d logistic(x)     ",
	"dot1d bloques         ",
	"dot1d mul             "
};

mdl_inst_f plume_inst[INSTS] = {
	plume_filtres_prixs,
	plume_filtres_prixs_tolerant,
	plume_dot1d_tanh,
	plume_dot1d_logistic,
	plume_dot1d_blk,
	plume_dot1d_mul
};

uint * UNIFORME_C(uint x) {
	uint * ret = alloc<uint>(C);
	FOR(0, i, C) ret[i] = x;
	return ret;
};

static void calculer_normalisee__et__dif_normalisee(Mdl_t * mdl) { 
	FOR(0, b, BLOQUES) {
		FOR(DEPART, t, FIN) {
			//	_max & _min pour ce filtre-8
			float _max, _min;
			
			if (mdl->type_de_norme[b] == NORME_CLASSIQUE) {
				_max = mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle];
				_min = mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle];
				FOR(1, i, N_FLTR) {
					float xi = mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle];
					if (_max < xi)
						_max = xi;
					if (_min > xi)
						_min = xi;
				}


			} else if (mdl->type_de_norme[b] == NORME_RELATIVE) {
				_max = fabs(mdl->bloque[b]->brute[t - 0*mdl->bloque[b]->intervalle]);
				FOR(1, i, N_FLTR) {
					float xi = fabs(mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle]);
					if (_max < xi)
						_max = xi;
				}
				_max = +_max;
				_min = -_max;

			} else if (mdl->type_de_norme[b] == NORME_THEORIQUE) {
				_min = /*1.0;//*/mdl->min_theorique[b];
				_max = /*0.0;//*/mdl->max_theorique[b];
				FOR(0, i, N_FLTR) {
					float xi = mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle];
					if (!(_min <= xi && xi <= _max))
						printf("%f  %f %f\n", xi, _min, _max);
					ASSERT(_min <= xi && xi <= _max);
				}

			} else {
				ERR("Norme ni classique, ni theorique, mdl->type_de_norme[b] == %i", mdl->type_de_norme[b]);
			};

			if (_min == _max) {
				FOR(0, i, N_FLTR) printf("%f\n", mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle]);

				ERR("_mi==_max (%f!=%f) b=%i type=%i", _min, _max, b, mdl->type_de_norme[b]);
				//ASSERT(_min != _max);
			}

			//
			FOR(0, i, N_FLTR) {
				mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] = ( mdl->bloque[b]->brute[t-i*mdl->bloque[b]->intervalle] - _min)/( _max - _min );
				//if (mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i]==0) printf("%i %i %i\n", b,t,i);
			}
		};

		//#pragma omp parallel
		//#pragma omp for
		FOR(DEPART, t, FIN) {
			FOR(1, i, N_FLTR)
				mdl->dif_normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] = mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i] - mdl->normalisee[b*PRIXS*N_FLTR+t*N_FLTR+i-1];
			mdl->dif_normalisee[b*PRIXS*N_FLTR+t*N_FLTR+N_FLTR+0] = 0.f;
		}
	}

	mdl->normalisee__d     = cpu_vers_gpu<float>(mdl->normalisee,     BLOQUES * PRIXS * N_FLTR);
	mdl->dif_normalisee__d = cpu_vers_gpu<float>(mdl->dif_normalisee, BLOQUES * PRIXS * N_FLTR);
};

static uint * tout_zeroiser = UNIFORME_C(1);

Mdl_t * cree_mdl(
	uint T,
	uint Y[C], uint insts[C],
	uint cible[C], uint decale_future[C],
	ema_int_t * bloque[BLOQUES]
) {
	ASSERT(Y[C-1] == P);
	ASSERT(Y[ 0 ] == BLOQUES * F_PAR_BLOQUES);
	ASSERT(insts[C-1] == DOT1D);				//	Afin d'assurer un Y=inst_VARS
	
	Mdl_t * mdl = alloc<Mdl_t>(1);

	mdl->T = T;

	//
	FOR(0, i, BLOQUES) {
		mdl->bloque[i]  = bloque[i];
		mdl->intervalles[i] = bloque[i]->intervalle;
		//
		mdl->type_de_norme[i] = bloque[i]->type_de_norme;
		mdl->min_theorique[i] = bloque[i]->min_theorique;
		mdl->max_theorique[i] = bloque[i]->max_theorique;
	};

	mdl->intervalles__d = cpu_vers_gpu<uint>(mdl->intervalles, BLOQUES);

	//
	calculer_normalisee__et__dif_normalisee(mdl);
	//raise(SIGINT);

	//	Architecture
	memcpy(mdl->insts,                 insts, sizeof(uint) * C);
	memcpy(mdl->Y,                         Y, sizeof(uint) * C);
	memcpy(mdl->cible,                 cible, sizeof(uint) * C);
	memcpy(mdl->decale_future, decale_future, sizeof(uint) * C);

	//	Allocation
	mdl->total_POIDS = 0;
	FOR(0, c, C) {
		//if (c>0) ASSERT(insts[c] != 0);
		ASSERT(Y[c] <= MAX_Y);
		//
		ASSERT(cible[c] == CIBLE_NORMALE || cible[c] == CIBLE_FILTRES_FUTURES);
		if (cible[c] == CIBLE_FILTRES_FUTURES) {
			ASSERT(Y[c] == F_PAR_BLOQUES * BLOQUES);
		}
		ASSERT(decale_future[c] <= MAX_DECALE_FUTURE);
		//
		cree_inst[insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		mdl->y [c] = alloc<float>(mdl->inst_VARS [c] * T);
		mdl->l [c] = alloc<float>(mdl->inst_LOCDS[c] * T);
		mdl->dy[c] = alloc<float>(mdl->inst_VARS [c] * T);
		mdl->dp[c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		mdl->y__d [c] = cudalloc<float>(mdl->inst_VARS [c] * T);
		mdl->l__d [c] = cudalloc<float>(mdl->inst_LOCDS[c] * T);
		mdl->dy__d[c] = cudalloc<float>(mdl->inst_VARS [c] * T);
		mdl->dp__d[c] = cudalloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];

		//	-----
		if (cible[c] == CIBLE_FILTRES_FUTURES) {
			mdl->constantes   [c] = gpu_vers_cpu<float>(mdl->p__d[c], mdl->inst_POIDS[c]);
			mdl->constantes__d[c] = cpu_vers_gpu<float>(mdl->p   [c], mdl->inst_POIDS[c]);
		}
	}
	ASSERT(mdl->inst_DEPART_SORTIE[C-1] == 0);
	//
	mdl_normer_les_filtres(mdl);
	//
	//uint * tout_zeroiser = UNIFORME_C(1);
	//
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	mdl_zero_deriv_cpu(mdl, tout_zeroiser);
	//
	return mdl;
};

void mdl_re_cree_poids(Mdl_t * mdl) {
	//	Allocation
	mdl->total_POIDS = 0;
	FOR(0, c, C) {
		if (c>0) ASSERT(mdl->insts[c] != 0);
		ASSERT(mdl->Y[c] <= MAX_Y);
		//
		free(mdl->p[c]);
		CONTROLE_CUDA(hipFree(mdl->p__d[c]));
		//
		cree_inst[mdl->insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];
	}
};

void mdl_changer_couche_Y(Mdl_t * mdl, uint c, uint nouveau_Y) {
	mdl->total_POIDS -= mdl->inst_POIDS[c];
	{
		if (c>0) ASSERT(mdl->insts[c] != 0);
		mdl->Y[c] = nouveau_Y;
		ASSERT(mdl->Y[c] <= MAX_Y);
		//
		free(mdl->p[c]);
		CONTROLE_CUDA(hipFree(mdl->p__d[c]));
		//
		cree_inst[mdl->insts[c]](mdl, c);
		//
		//mdl->p [c] = alloc<float>(mdl->inst_POIDS[c]);
		//
		mdl->p__d [c] = cpu_vers_gpu<float>(mdl->p[c], mdl->inst_POIDS[c]);
		//
		mdl->total_POIDS += mdl->inst_POIDS[c];
	}
};

void mdl_normer_les_filtres(Mdl_t * mdl) {
	FOR(0, b, BLOQUES) {
		FOR(0, f, F_PAR_BLOQUES) {
			float min, max;
			uint type_norme = mdl->type_de_norme[b];
			if (type_norme == NORME_CLASSIQUE) {
				max=mdl->p[0][b*F_PAR_BLOQUES*N + f*N+0];
				min=mdl->p[0][b*F_PAR_BLOQUES*N + f*N+0];
				FOR(1, i, N) {
					if (max < mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]) max = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
					if (min > mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]) min = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
				}
			} else if (type_norme == NORME_THEORIQUE || type_norme == NORME_RELATIVE) {
				min = 0.0;	//se sont des filtres
				max = 1.0;	//pas les natures

				//	Borne au cas ou
				FOR(0, i, N) {
					float f_val = mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i];
					mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i] = MAX2(MIN2(f_val, max), min);
				}
			} else {
				ERR("mdl->type_de_norme[b]=%i", mdl->type_de_norme[b]);
			}
			//
			FOR(0, i, N) mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i] = (mdl->p[0][b*F_PAR_BLOQUES*N + f*N+i]-min)/(max-min);
		};
	}
	CONTROLE_CUDA(hipMemcpy(mdl->p__d[0], mdl->p[0], sizeof(float)*BLOQUES*F_PAR_BLOQUES*N, hipMemcpyHostToDevice))
};

PAS_OPTIMISER()
void mdl_verif(Mdl_t * mdl) {
	FOR(1, c, C) {
		float * r = gpu_vers_cpu<float>(mdl->p__d[c], mdl->inst_POIDS[c]);
		FOR(0, i, mdl->inst_POIDS[c]) ASSERT(fabs(r[i]-mdl->p[c][i]) < 0.01);
		free(r);
	}
};

PAS_OPTIMISER()
void mdl_poids_cpu_vers_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],  mdl->p[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void mdl_poids_gpu_vers_cpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p[c],  mdl->p__d[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyDeviceToHost));
	}
};

PAS_OPTIMISER()
void mdl_gpu_vers_cpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p[c],  mdl->p__d[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->y[c],  mdl->y__d[c],  sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->l[c],  mdl->l__d[c],  sizeof(float)*mdl->inst_LOCDS[c]*mdl->T, hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dy[c], mdl->dy__d[c], sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dp[c], mdl->dp__d[c], sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyDeviceToHost));
	}
}

PAS_OPTIMISER()
void mdl_cpu_vers_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],  mdl->p[c],  sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->y__d[c],  mdl->y[c],  sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->l__d[c],  mdl->l[c],  sizeof(float)*mdl->inst_LOCDS[c]*mdl->T, hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dy__d[c], mdl->dy[c], sizeof(float)*mdl->inst_VARS[c]*mdl->T,  hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dp__d[c], mdl->dp[c], sizeof(float)*mdl->inst_POIDS[c],       hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void liberer_mdl(Mdl_t * mdl) {
	FOR(0, c, C) {
		free(mdl->p [c]);
		free(mdl->y [c]);
		free(mdl->l [c]);
		free(mdl->dy[c]);
		free(mdl->dp[c]);
		//
		CONTROLE_CUDA(hipFree(mdl->p__d [c]));
		CONTROLE_CUDA(hipFree(mdl->y__d [c]));
		CONTROLE_CUDA(hipFree(mdl->l__d [c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dp__d[c]));
	}
};

PAS_OPTIMISER()
void mdl_zero_cpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		memset(mdl->y [c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T);
	}
};

PAS_OPTIMISER()
void mdl_zero_gpu(Mdl_t * mdl) {
	FOR(0, c, C) {
		CONTROLE_CUDA(hipMemset(mdl->y__d [c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_cpu(Mdl_t * mdl, uint zeroiser[C]) {
	FOR(0, c, C) {
		if (zeroiser[c]) {
			memset(mdl->dy[c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T);
			memset(mdl->dp[c], 0, sizeof(float) * mdl->inst_POIDS[c]);
		}
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_gpu(Mdl_t * mdl, uint zeroiser[C]) {
	FOR(0, c, C) {
		if (zeroiser[c]) {
			CONTROLE_CUDA(hipMemset(mdl->dy__d[c], 0, sizeof(float) * mdl->inst_VARS [c] * mdl->T));
			CONTROLE_CUDA(hipMemset(mdl->dp__d[c], 0, sizeof(float) * mdl->inst_POIDS[c]));
		}
	}
};