#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

//	======================= Cible Decale Future =======================

static float filtre(float * x, float * dif_x, float * f) {
	float s = 0, d = 0;
	float f_nouveau = f[0];
	s += sqrtf(1 + fabs(x[0] - f_nouveau));
	float f_avant   = f_nouveau;
	FOR(1, i, N) {
		f_nouveau = f[i];
		s += sqrtf(1 + fabs(  x[i]   -   f_nouveau  ));
		d += powf((1 + fabs(dif_x[i] - (f_nouveau-f_avant))), 2);
		f_avant   = f_nouveau;
	};

	s = s/(float)(N)-1;
	d = d/(float)(N-1)-1;

	float y = expf(-s*s -d*d);

	return 2*y-1;
};

void intel_score_cible_filtres_future(
	uint DECALE_FUTURE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * dy)
{
	FOR(0, t, T) {
		FOR(0, b, bloques) {
			FOR(0, _f, f_par_bloque) {
				float flt = filtre(
						x + b*PRIXS*N_FLTR + (depart+t+DECALE_FUTURE)*N_FLTR,
					dif_x + b*PRIXS*N_FLTR + (depart+t+DECALE_FUTURE)*N_FLTR,
					f     + b*f_par_bloque*N     + _f*N
				);

				//	On ecrit par dessus le score
				dy[(0+t)*bloques*f_par_bloque + b*f_par_bloque + _f] = \
					(y[(0+t)*bloques*f_par_bloque + b*f_par_bloque + _f] - flt)/10000;
			}
		}
	}
};

#define BLOQUE_T  16

#define _repete_T 16

static __global__ void kerd_score_cible_filtres_futureshared(
	uint DECALE_FUTURE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * dy)
{
	uint depart_bloque_t = blockIdx.y * BLOQUE_T * _repete_T;
	uint depart_thread_t = depart_bloque_t + threadIdx.y * _repete_T;

	uint _b = blockIdx.x;
	uint _f = blockIdx.z;	//(ligne dans bloque)

	uint LIGNE  = _b;
	uint BLOQUE = _b; 

	uint thx = threadIdx.x;
	uint thy__t = threadIdx.y;

	//if (_t < T)
	__shared__ float __f__[N];
	//
	if (thy__t==0) __f__[thx]  = f[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx];
	__syncthreads();
	//
	float fi, fi1;
	fi = __f__[thx];
	if (thx != 0)
		fi1 = __f__[thx-1];
	//
	__shared__ float __ret[BLOQUE_T][2];	//s, d
	__shared__ float __y  [BLOQUE_T];
	//
	float xi, dif_xi;
	//
	uint _t;
	FOR(0, plus_t, _repete_T) {
		_t = depart_thread_t + plus_t;
		//
		if (thx < 2) {
			__ret[thy__t][thx] = 0;
		}
		__syncthreads();
		//
		xi = x[LIGNE*PRIXS*N_FLTR + (depart+_t+DECALE_FUTURE)*N_FLTR + thx];
		//
		if (thx != 0) {
			dif_xi = dif_x[LIGNE*PRIXS*N_FLTR + (depart+_t+DECALE_FUTURE)*N_FLTR + thx];
			atomicAdd(&__ret[thy__t][1], powf((1 + fabs(dif_xi - (fi-fi1))), 2));
		}
		atomicAdd(&__ret[thy__t][0], sqrtf(1 + fabs(xi - fi)));
		__syncthreads();
		//
		if (thx < 2) {
			__ret[thy__t][thx] = __ret[thy__t][thx]/(float)(8-thx) - 1.0;
		}
		__syncthreads();
		//
		if (thx < 1) {
			__y[thy__t] = expf(-__ret[thy__t][0]*__ret[thy__t][0] -__ret[thy__t][1]*__ret[thy__t][1]);
		}
		__syncthreads();
		//
		/*if (thx < 2) {
			locd[(0+_t)*BLOQUES*(F_PAR_BLOQUES*2) + BLOQUE*(F_PAR_BLOQUES*2) + _f*2 + thx] = -2*2*__ret[thy__t][thx]*__y[thy__t];
		}
		__syncthreads();*/
		//
		if (thx < 1) {
			float flt = 2*__y[thy__t] - 1;
			float d_score = y[(0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f] - flt;
			dy[(0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f] = d_score / 10000;
		}
	};
};

void d_nvidia_score_cible_filtres_futureshared(
	uint DECALE_FUTURE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * dy)
{
	ASSERT(BLOQUE_T*_repete_T <= T);
	kerd_score_cible_filtres_futureshared<<<dim3(bloques, KERD((DIV(T,_repete_T)), BLOQUE_T), F_PAR_BLOQUES), dim3(N, BLOQUE_T,1)>>>(
		DECALE_FUTURE,
		X_vars, Y_vars,
		depart, T,
		bloques,
		x, dif_x,
		f,
		y,
		dy);
	ATTENDRE_CUDA();
};

//	===================================================================

void mdl_df(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	RETRO_FOR(0, c, C) {
		if        (mdl->cible[c] == CIBLE_NORMALE        ) {
			//	Rien a faire
		} else if (mdl->cible[c] == CIBLE_FILTRES_FUTURES) {
			uint depart = t0;
			uint X_vars=0, Y_vars=mdl->inst_VARS[c];
			uint T = (t1-t0);
			ASSERT(T == mdl->T);
			if (mode == 0) {
				intel_score_cible_filtres_future(
					mdl->decale_future[c],
					X_vars, Y_vars,
					depart, T,
					BLOQUES, F_PAR_BLOQUES,
					mdl->normalisee, mdl->dif_normalisee,
					mdl->constantes[c],//mdl->p[c],
					mdl->y[c],
					mdl->dy[c]);
			} else {
				d_nvidia_score_cible_filtres_futureshared(
					mdl->decale_future[c],
					X_vars, Y_vars,
					depart, T,
					BLOQUES, F_PAR_BLOQUES,
					mdl->normalisee__d, mdl->dif_normalisee__d,
					mdl->constantes__d[c],//mdl->p__d [c],
					mdl->y__d [c],
					mdl->dy__d[c]);
			};
		} else {
			ERR("Pas de cible %i", mdl->cible[c]);
		}
		//
		inst_df[mdl->insts[c]](mdl, c, mode, t0, t1, _t_MODE, GRAINE);
	};
};