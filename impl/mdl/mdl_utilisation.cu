#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_les_gains(Mdl_t * mdl, uint t0, uint t1, uint mode, float GRAND_COEF, uint _t_MODE, uint GRAINE) {
	ASSERT(GRAND_COEF >= 2);
	ASSERT(mdl->T == (t1-t0));
	float * _y = gpu_vers_cpu<float>(mdl->y__d[C-1], (t1-t0)*P);
	float somme = 0;
	float potentiel = 0;
	FOR(t0, t, t1) {
		uint depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, t0, DEPART, FIN, (t-t0));
		//
		somme     += powf(fabs(prixs[/*t*/depart_plus_t+1]/prixs[/*t*/depart_plus_t]-1),GRAND_COEF) * (signe((prixs[/*t*/depart_plus_t+1]/prixs[/*t*/depart_plus_t]-1)) == signe(_y[(t-t0)*P+0]));
		potentiel += powf(fabs(prixs[/*t*/depart_plus_t+1]/prixs[/*t*/depart_plus_t]-1),GRAND_COEF);
	}
	free(_y);
	return somme / potentiel;
};

float mdl_score(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1],    t0, (t1-t0), _t_MODE, GRAINE);
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	//
	if (mode == 0) return  intel_score_finale(somme_score, (t1-t0), _t_MODE, GRAINE);
	else           return nvidia_score_finale(somme_score, (t1-t0), _t_MODE, GRAINE);
};

float* mdl_pred(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	if (mode == 0) return  intel_prediction(mdl->y[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	else           return nvidia_prediction(mdl->y__d[C-1], t0, (t1-t0), _t_MODE, GRAINE);
};

void mdl_aller_retour(Mdl_t * mdl, uint t0, uint t1, uint mode, uint _t_MODE, uint GRAINE) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	mdl_f(mdl, t0, t1, mode, _t_MODE, GRAINE);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	//
	float d_score;
	if (mode == 0) d_score =  d_intel_score_finale(somme_score, (t1-t0), _t_MODE, GRAINE);
	else           d_score = d_nvidia_score_finale(somme_score, (t1-t0), _t_MODE, GRAINE);
	//
	if (mode == 0)  d_intel_somme_score(d_score, mdl->y[C-1],    mdl->dy[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	else           d_nvidia_somme_score(d_score, mdl->y__d[C-1], mdl->dy__d[C-1], t0, (t1-t0), _t_MODE, GRAINE);
	mdl_df(mdl, t0, t1, mode, _t_MODE, GRAINE);
};