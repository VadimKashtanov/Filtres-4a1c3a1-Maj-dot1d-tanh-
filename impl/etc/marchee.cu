#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

char * nom_sources[SOURCES] = {
	" prixs ",
	"volumes",
	"  haut ",
	"  bas  ",
	" median"
};

//	Sources
float   prixs[PRIXS] = {};
float volumes[PRIXS] = {};
float    high[PRIXS] = {};
float     low[PRIXS] = {};
float  median[PRIXS] = {};

float *   prixs__d = 0x0;
float * volumes__d = 0x0;
float *    high__d = 0x0;
float *     low__d = 0x0;
float *  median__d = 0x0;

float * sources[SOURCES] = {
	prixs, volumes, high, low, median
};

float * sources__d[SOURCES] = {
	prixs__d, volumes__d, high__d, low__d, median__d
};

void charger_les_prixs() {
	uint __PRIXS;
	FILE * fp;
	//
	fp = fopen("prixs/prixs.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(prixs, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/volumes.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(volumes, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/high.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(high, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/low.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(low, sizeof(float), PRIXS, fp);
	fclose(fp);
	//
	fp = fopen("prixs/median.bin", "rb");
	ASSERT(fp != 0);
	(void)!fread(&__PRIXS, sizeof(uint), 1, fp);
	ASSERT(__PRIXS == PRIXS);
	(void)!fread(median, sizeof(float), PRIXS, fp);
	fclose(fp);
};

//	===========================================================

void ema_int_calc_ema(ema_int_t * ema_int) {
	//			-- Parametres --
	uint K = ema_int->K_ema;
	float _K = 1.0 / ((float)K);
	//	EMA
	ema_int->ema[0] = sources[ema_int->source][0];
	FOR(1, i, PRIXS) {
		ema_int->ema[i] = ema_int->ema[i-1] * (1.0 - _K) + sources[ema_int->source][i]*_K;
	}
};

//	===========================================================

uint nature_multiple_interv[NATURES] = {
	0,
	0,
	0,
	14,
	14
};

nature_f fonctions_nature[NATURES] = {
	nature0__direct,
	nature1__macd,
	nature2__chiffre,
	nature3__awesome,
	nature4__pourcent_r,
	nature5__rsi,
};

uint NATURE_PARAMS[NATURES] = {
	0,
	1,
	1,
	1,
	2,
	2
};

uint min_param[NATURES][MAX_PARAMS] = {
	{0,0,0,0},
	{1,0,0,0},
	{1,0,0,0},
	{1,0,0,0},
	{1,1,0,0},
	{1,1,0,0}
};

uint max_param[NATURES][MAX_PARAMS] = {
	{0,                0,       0,        0      }, 
	{MAX_COEF_MACD,    0,       0,        0      },
	{MAX_CHIFFRE,      0,       0,        0      },
	{MAX_COEF_AWESOME, 0,       0,        0      },
	{MAX_INTERVALLE,   MAX_EMA, 0,        0      },
	{MAX_INTERVALLE,   MAX_EMA, 0,        0      } 
};

char * nom_natures[NATURES] {
	"directe",
	"  macd ",
	"chiffre",
	"awesome",
	"  %R   ",
	"  RSI  "
};

ema_int_t * cree_ligne(uint source, uint nature, uint K_ema, uint intervalle, uint params[MAX_PARAMS]) {
	ema_int_t * ret = alloc<ema_int_t>(1);
	//
	ret->source = source;
	ret->nature = nature;
	ret->K_ema  = K_ema;
	ret->intervalle = intervalle;
	//
	ASSERT(intervalle <= MAX_INTERVALLE);
	ASSERT(K_ema      <= MAX_EMA);
	//
	memcpy(ret->params, params, sizeof(uint) * MAX_PARAMS);
	//
	ema_int_calc_ema(ret);
	fonctions_nature[nature](ret);
	//
	return ret;
};

void liberer_ligne(ema_int_t * ema_int) {

};

void charger_vram_nvidia() {
	CONTROLE_CUDA(hipMalloc((void**)&  prixs__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&volumes__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&   high__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)&    low__d, sizeof(float) * PRIXS));
	CONTROLE_CUDA(hipMalloc((void**)& median__d, sizeof(float) * PRIXS));
	//
	CONTROLE_CUDA(hipMemcpy(  prixs__d,   prixs, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(volumes__d, volumes, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(   high__d,    high, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(    low__d,     low, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy( median__d,  median, sizeof(float) * PRIXS, hipMemcpyHostToDevice));
};

void     liberer_cudamalloc() {
	CONTROLE_CUDA(hipFree(  prixs__d));
	CONTROLE_CUDA(hipFree(volumes__d));
	CONTROLE_CUDA(hipFree(   high__d));
	CONTROLE_CUDA(hipFree(    low__d));
	CONTROLE_CUDA(hipFree( median__d));
};

void charger_tout() {
	//	Assertions
	FOR(0, i, NATURES) ASSERT(nature_multiple_interv[i] <= MAX_MULTPLE_INTERV_NATURES);
	//
	printf("charger_les_prixs : ");    MESURER(charger_les_prixs());
	printf("charger_vram_nvidia : ");  MESURER(charger_vram_nvidia());
};

void liberer_tout() {
	titre("Liberer tout");
	liberer_cudamalloc();
};

ema_int_t * lire_ema_int(FILE * fp) {
	uint source, nature, K_ema, intervalle;
	uint params[MAX_PARAMS];
	FREAD(&source,     sizeof(uint), 1, fp);
	FREAD(&nature,     sizeof(uint), 1, fp);
	FREAD(&K_ema,      sizeof(uint), 1, fp);
	FREAD(&intervalle, sizeof(uint), 1, fp);
	//
	FREAD(&params,     sizeof(uint), MAX_PARAMS, fp);
	//
	return cree_ligne(source, nature, K_ema, intervalle, params);
};

void      ecrire_ema_int(ema_int_t * ema_int, FILE * fp) {
	FWRITE(&ema_int->source,     sizeof(uint), 1, fp);
	FWRITE(&ema_int->nature,     sizeof(uint), 1, fp);
	FWRITE(&ema_int->K_ema,      sizeof(uint), 1, fp);
	FWRITE(&ema_int->intervalle, sizeof(uint), 1, fp);
	//
	FWRITE(&ema_int->params,     sizeof(uint), MAX_PARAMS, fp);
};

char * nom_type_de_norme[3] = {
	"NORME_CLASSIQUE",
	"NORME_THEORIQUE",
	"NORME_RELATIVE "
};