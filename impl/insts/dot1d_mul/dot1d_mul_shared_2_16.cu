#include "hip/hip_runtime.h"
#include "dot1d.cuh"

/*	Difference :
	Au lieux de directement deriver avec que des atomicAdd le
__shared__ noyau, on fait la méthode que j'avais avant
ou on fait une autre opération pour calc dx et dp.

	Mathématiquement ca correspond a deriver y=X@P+B
en dX=p@dY.T
	dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	dp = x.T @ ((y-_y)*dtanh(x@p))
*/

#define BLOQUE 16
#define BLOQUE_MAX 16

static __global__ void kerd_mul_stricte_16__shared2(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	/*// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, X/BLOQUE) {
		__partage__x[thy][thx] = x[(depart+_t)*( X_vars ) + DEPART_x +d*BLOQUE + thx];
		__partage__p[thy][thx] = p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

#define __partage__b __partage__x[0]

	if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = ACTIV(ACTIVATION, s);
	   y[(depart+_t)*Y + _y] = a;
	locd[(depart+_t)*Y + _y] = dACTIV(ACTIVATION, s,a);*/
};

void nvidia_dot1d_mul_shared_2_16(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	/*if (T%BLOQUE!=0) ERR("ATTENTION T%%16 != 0 (T=%i)", T);
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_mul_stricte_16__shared2<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
		ATTENDRE_CUDA();
	} else {
		ERR("Impossible");
	}*/
}

static __global__ void kerd_mul_stricte_16__shared2____dx(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	/*//dx = (p @ ((y-_y)*dtanh(x@p)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	//uint _y = thx + blockIdx.x * blockDim.x;
	uint _x = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, Y/BLOQUE) {
		__partage__x[thy][thx] = locd[(depart+_t)*Y+d*BLOQUE+thx] * dy[(depart+_t)*Y+d*BLOQUE+thx];//x[(depart+_t)*( X ) + d*BLOQUE + thx];
		__partage__p[thy][thx] = p[(d*BLOQUE+thy)*(X+1) + _x];//p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	dx[(depart+_t)*X_vars+DEPART_x +_x]   = s;
	//printf("s=%f\n", s);*/
};


static __global__ void kerd_mul_stricte_32__shared2____dp(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{

	/*//dp = x.T @ ((y-_y)*dtanh(x@p))

	__shared__ float __partage__x[BLOQUE_MAX][BLOQUE_MAX];
	__shared__ float __partage__p[BLOQUE_MAX][BLOQUE_MAX];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	uint _y = thy + blockIdx.y * blockDim.y;

	float s = 0;
	float biais = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		__partage__x[thy][thx] = locd[(depart+d*BLOQUE_MAX+thx)*Y+_y] * dy[(depart+d*BLOQUE_MAX+thx)*Y+_y];//x[(depart+_t)*( X ) + d*BLOQUE + thx];
		__partage__p[thy][thx] = x[(depart+(d*BLOQUE_MAX+thy))*X_vars+DEPART_x +_x];//p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE_MAX) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
			if (_x == 0) biais += __partage__x[thy][i];
		}
		__syncthreads();
	//};

#define __partage__b __partage__x[0]

	//if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	if (_x == 0) atomicAdd(&dp[_y*(X+1) + (X+1-1)], biais);
	__syncthreads();

	atomicAdd(&dp[_y*(X+1)+_x], s);*/
};

void d_nvidia_dot1d_mul_shared_2_16(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	/*if (T%(MAX2(BLOQUE_MAX,BLOQUE))!=0) ERR("ATTENTION T%%%i != 0 (T=%i)", T, (MAX2(BLOQUE_MAX,BLOQUE)));
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_mul_stricte_16__shared2____dx<<<dim3(KERD(X, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		kerd_mul_stricte_32__shared2____dp<<<dim3(KERD(X, BLOQUE_MAX), KERD(Y, BLOQUE_MAX), DIV(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX,1)>>>(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		ATTENDRE_CUDA();
	} else {
		ERR("Impossible");
	}*/
}