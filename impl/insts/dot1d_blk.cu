#include "hip/hip_runtime.h"
#include "dot1d_blk.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_dot1d_blk(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	//
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	mdl->inst_POIDS        [c] = P_blk*DOT1D_BLK_BLOQUES;
	mdl->inst_VARS         [c] = mdl->Y[c];
	mdl->inst_LOCDS        [c] = mdl->Y[c];
	mdl->inst_SORTIES      [c] = mdl->Y[c];
	mdl->inst_DEPART_SORTIE[c] = mdl->Y[c] - mdl->Y[c];
	//
	printf("Poids = %i\n", P_blk*DOT1D_BLK_BLOQUES);
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);

	/*			Distribution Uniforme et Normale	 	*/

	//	somme(r) = 1 && |max(r)|==|min(r)|==COEF
	/*FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
		//uint depart_y = dot1d_blk * Y_blk;
		//uint depart_x = dot1d_blk * X_blk;
		uint depart_p = dot1d_blk * P_blk;
		//
		FOR(0, y, Y_blk) {
			FOR(0, x, X_blk+1) {
				mdl->p[c][depart_p+y*(X_blk+1)+x] = (2*rnd()-1) * sqrtf(15.0 / X_blk);
			}
		}
	}*/
	FOR(0, i, mdl->inst_POIDS[c]) {
		mdl->p[c][i] = (2*rnd()-1) * sqrtf(/*10.0*/6.0 / (X_blk+Y_blk));
	}
};

void plume_dot1d_blk(Mdl_t * mdl, uint c)
{
	printf("POIDS dot1d_blk: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	//
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
		printf("[bloque=%i] ", dot1d_blk);
		//uint depart_y = dot1d_blk * Y_blk;
		//uint depart_x = dot1d_blk * X_blk;
		uint depart_p = dot1d_blk * P_blk;
		//
		FOR(0, y, Y_blk) {
			printf("y=%i : ", y);
			FOR(0, x, X_blk) {
				printf("%+f,", mdl->p[c][depart_p+y*(X_blk+1)+x]);
			}
			printf(" biais=%+f\n", mdl->p[c][depart_p+y*(X_blk+1)+X_blk+1-1]);
		}
	}
};

void intel_dot1d_blk(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
//#pragma omp parallel
//#pragma omp for
	FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
		uint depart_y = dot1d_blk * Y_blk;
		uint depart_x = dot1d_blk * X_blk;
		uint depart_p = dot1d_blk * P_blk;
		//
		/*#pragma omp parallel
		#pragma omp for*/
		FOR(0, t, T) {
			FOR(0, _y, Y_blk) {
				float s = p[depart_p+_y*(X_blk+1)+(X_blk+1-1)];
				FOR(0, k, X_blk) {
					float __x = x[(depart+t)*X_vars+DEPART_x+depart_x+k];
					float __p = p[depart_p+_y*(X_blk+1)+k];
					s += __x * __p;
				}
				float a = dot1d_blk_ACTIV(dot1d_blk_ACTIVATION, s);
				y[(depart+t)*Y+depart_y+_y]    = a;
				locd[(depart+t)*Y+depart_y+_y] = dot1d_blk_dACTIV(dot1d_blk_ACTIVATION, s, a);
			}
		}
	}
}

void d_intel_dot1d_blk(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,	//<-- eventuallement pour un 
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	/*uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
		uint depart_y = dot1d_blk * Y_blk;
		uint depart_x = dot1d_blk * X_blk;
		uint depart_p = dot1d_blk * P_blk;
		//
		FOR(0, t, T) {
			FOR(0, _y, Y_blk) {
				float ds = locd[(depart+t)*Y+depart_y+_y] * dy[(depart+t)*Y+depart_y+_y];
				//
				dp[depart_p+_y*(X_blk+1)+(X_blk+1-1)] += ds;
				FOR(0, k, X_blk) {
					float __x = x[(depart+t)*X_vars+DEPART_x+depart_x+k];
					float __p = p[depart_p+_y*(X_blk+1)+k];
					//s += __x * __p;
					dx[(depart+t)*X_vars+DEPART_x+depart_x+k] += ds * __p; 
					dp[depart_p+_y*(X_blk+1)+k] += ds * __x;
				}
			}
		}
	}*/
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
		uint depart_y = dot1d_blk * Y_blk;	//depat d'un bloque en x
		uint depart_x = dot1d_blk * X_blk;	// 					en y
		uint depart_p = dot1d_blk * P_blk;	//					en p
		//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	//#pragma omp parallel
	//#pragma omp for
		FOR(0, t, T) {
			FOR(0, _x, X_blk) {
				//float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
				float s = 0;
				FOR(0, k, Y_blk) {
					float __x = p[depart_p+k*(X_blk+1)+_x];//x[(depart+t)*X+k];
					float __p = locd[(depart+t)*Y+depart_y+k] * dy[(depart+t)*Y+depart_y+k];//p[_y*(X+1)+k];
					s += __x * __p;
				}
				dx[(depart+t)*X_vars+DEPART_x+depart_x+_x]   = s;
			}
		}

		//dp = x.T @ ((y-_y)*dtanh(x@p))
	//#pragma omp parallel
	//#pragma omp for
		FOR(0, _y, Y_blk) {
			float dbiais = 0;
			FOR(0, _x, X_blk) {
				float s = 0;
				FOR(0, t, T) {
					float __x = locd[(depart+t)*Y+depart_y+_y] * dy[(depart+t)*Y+depart_y+_y];//x[(depart+t)*X+k];
					float __p = x[(depart+t)*X_vars+DEPART_x+depart_x+_x];//p[_y*(X+1)+k];
					s += __x * __p;
					if (_x == 0) {	//	Biais
						dbiais += __x;
					}
				}
				dp[depart_p+_y*(X_blk+1)+_x] = s;
			}
			dp[depart_p+_y*(X_blk+1)+(X_blk+1-1)] = dbiais;
		}
	}
}

//	=========================================================

void f_dot1d_blk(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	ERR("Pas ajouté le depart_plus_t")
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		intel_dot1d_blk(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst]);
	} else if (mode == 1) {
		nvidia_dot1d_blk_naive(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 2) {
		nvidia_dot1d_blk_shared_2_16(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 3) {
		nvidia_dot1d_blk_shared_2_16(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_dot1d_blk(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	ERR("Pas ajouté le depart_plus_t")
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		d_intel_dot1d_blk(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst],
			mdl->dy[inst],
			mdl->dy[inst-1],
			mdl->dp[inst]);
	} else if (mode == 1) {
		d_nvidia_dot1d_blk_naive(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == 2) {
		d_nvidia_dot1d_blk_shared_2_16(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == 3) {
		d_nvidia_dot1d_blk_shared_2_16(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}