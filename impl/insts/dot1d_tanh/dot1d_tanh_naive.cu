#include "hip/hip_runtime.h"
#include "dot1d_tanh.cuh"

#define BLOQUE_T 32
#define BLOQUE_Y 32

static __global__ void kerd_dot1d_tanh_naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float s = p[_y*(X+1) + (X+1-1)];
		FOR(0, i, X) s += x[(0+_t)*X_vars + DEPART_x + i] * p[_y*(X+1) + i];
		float a = ACTIV(ACTIVATION, s);
		y[/*(depart+_t)*/(0+_t)*Y + _y] = a;
		locd[/*(depart+_t)*/(0+_t)*Y + _y] = dACTIV(ACTIVATION, s,a);
	}
};

void nvidia_dot1d_tanh_naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	kerd_dot1d_tanh_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
}

//	============================= Derivation ==============================

static __global__ void kerd_deriv_dot1d_tanh_naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float _locd = locd[/*(depart+_t)*/(0+_t)*Y + _y] * dy[/*(depart+_t)*/(0+_t)*Y + _y];
		atomicAdd(&dp[_y*(X+1) + (X+1-1)], _locd);
		FOR(0, i, X) {
			atomicAdd(&dx[(0+_t)*X_vars + DEPART_x + i], _locd * p[_y*(X+1) + i]);
			atomicAdd(&dp[_y*(X+1) + i], _locd * x[(0+_t)*X_vars + DEPART_x + i]);
		}
	}
};

void d_nvidia_dot1d_tanh_naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	kerd_deriv_dot1d_tanh_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};