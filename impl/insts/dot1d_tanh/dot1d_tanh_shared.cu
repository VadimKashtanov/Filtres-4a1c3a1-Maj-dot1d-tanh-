#include "hip/hip_runtime.h"
#include "dot1d_tanh.cuh"

#define BLOQUE 16

static __global__ void kerd_stricte(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, X/BLOQUE) {
		__partage__x[thy][thx] = x[(depart+_t)*(X_vars) + DEPART_x+ d*BLOQUE + thx];
		__partage__p[thy][thx] = p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

#define __partage__b __partage__x[0]

	if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = ACTIV(ACTIVATION, s);
	   y[/*(depart+_t)*/(0+_t)*Y + _y] = a;
	locd[/*(depart+_t)*/(0+_t)*Y + _y] = dACTIV(ACTIVATION, s,a);
};

void nvidia_dot1d_tanh_shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_stricte<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
		ATTENDRE_CUDA();
	} else {
		nvidia_dot1d_tanh_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
	}
}

//	============================= Derivation ==============================

static __global__ void kerd_stricte_deriv(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	__shared__ float __partage__dx[BLOQUE][BLOQUE];
	__shared__ float __partage__dp[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float _locd = locd[(0+_t)*Y + _y] * dy[(0+_t)*Y + _y];

	atomicAdd(&dp[_y*(X+1) + (X)], _locd);

	FOR(0, d, X/BLOQUE) {
		__partage__x[thy][thx] = x[(0+_t)*(X_vars) + DEPART_x + d*BLOQUE + thx];
		__partage__p[thy][thx] = p[_y*(X+1) + d*BLOQUE + thy];
		__partage__dx[thy][thx] = 0;
		__partage__dp[thy][thx] = 0;
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) {
			//s += __partage__x[thy][i] * __partage__p[i][thx];
			atomicAdd(&__partage__dx[thy][i], _locd * __partage__p[i][thx]);
			atomicAdd(&__partage__dp[i][thx], _locd * __partage__x[thy][i]);
		}
		__syncthreads();

		atomicAdd(&dx[(0+_t)*(X_vars) + DEPART_x + d*BLOQUE + thx], __partage__dx[thy][thx]); 
		atomicAdd(&dp[_y*(X+1) + d*BLOQUE + thy], __partage__dp[thy][thx]);
	}
};

void d_nvidia_dot1d_tanh_shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	if (X%BLOQUE==0 && Y%BLOQUE==0 && T%BLOQUE==0) {
		kerd_stricte_deriv<<<dim3(KERD(Y, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
		ATTENDRE_CUDA();
	} else {
		d_nvidia_dot1d_tanh_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	}
}