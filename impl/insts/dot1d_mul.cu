#include "hip/hip_runtime.h"
#include "dot1d_mul.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define POLYNOMES 3

//y = x*logistic(ax+b)

void cree_dot1d_mul(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	mdl->inst_POIDS        [c] = (mdl->Y[c-1]+1)*mdl->Y[c] * POLYNOMES;
	mdl->inst_VARS         [c] = mdl->Y[c];
	mdl->inst_LOCDS        [c] = mdl->Y[c];
	mdl->inst_SORTIES      [c] = mdl->Y[c];
	mdl->inst_DEPART_SORTIE[c] = mdl->Y[c] - mdl->Y[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);
	//
	FOR(0, pol, POLYNOMES) {
		FOR(0, y, Y) {
			FOR(0, x, X+1) {
				mdl->p[c][pol*((X+1)*Y) + y*(X+1)+x] = (2*rnd()-1) * sqrtf(/*10.0*/ 10.0 / (X+Y));
			}
		}
	}
};

void plume_dot1d_mul(Mdl_t * mdl, uint c)
{
	printf("POIDS DOT1D_MUL (tanh(ax+b)*logistique(cx+d)+tanh(ex+f)): \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	printf("tanh(ax+b)\n");
	FOR(0, y, Y) {
		printf("y=%i : ", y);
		FOR(0, x, X) {
			printf("%+f,", mdl->p[c][0*(X+1)*Y + y*(X+1)+x]);
		}
		printf(" biais=%+f\n", mdl->p[c][0*(X+1)*Y + y*(X+1)+X+1-1]);
	}
	printf("logistique(cx+d)\n");
	FOR(0, y, Y) {
		printf("y=%i : ", y);
		FOR(0, x, X) {
			printf("%+f,", mdl->p[c][1*(X+1)*Y + y*(X+1)+x]);
		}
		printf(" biais=%+f\n", mdl->p[c][1*(X+1)*Y + y*(X+1)+X+1-1]);
	}
	printf("tanh(ex+f)\n");
	FOR(0, y, Y) {
		printf("y=%i : ", y);
		FOR(0, x, X) {
			printf("%+f,", mdl->p[c][2*(X+1)*Y + y*(X+1)+x]);
		}
		printf(" biais=%+f\n", mdl->p[c][2*(X+1)*Y + y*(X+1)+X+1-1]);
	}
};

void intel_dot1d_mul(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	float s;
	FOR(0, t, T) {
		FOR(0, _y, Y) {
			//	tanh(ax+b)
			s = p[0*(X+1)*Y + _y*(X+1)+(X+1-1)];
			FOR(0, k, X) {
				float __x = x[(depart+t)*X_vars+DEPART_x+k];
				float __p = p[0*(X+1)*Y + _y*(X+1)+k];
				s += __x * __p;
			}
			float P0 = mul_TANH(s);
			locd[(depart+t)*(Y*3)+_y+0] = P0;
		
			//	logistique(cx+d)
			s = p[1*(X+1)*Y + _y*(X+1)+(X+1-1)];
			FOR(0, k, X) {
				float __x = x[(depart+t)*X_vars+DEPART_x+k];
				float __p = p[1*(X+1)*Y + _y*(X+1)+k];
				s += __x * __p;
			}
			float P1 = mul_LOGISTIQUE(s);
			locd[(depart+t)*(Y*3)+_y+1] = P1;
		
			//	tanh(ex+f)
			s = p[2*(X+1)*Y + _y*(X+1)+(X+1-1)];
			FOR(0, k, X) {
				float __x = x[(depart+t)*X_vars+DEPART_x+k];
				float __p = p[2*(X+1)*Y + _y*(X+1)+k];
				s += __x * __p;
			}
			float P2 = mul_TANH(s);
			locd[(depart+t)*(Y*3)+_y+2] = P2;

			//	P0*P1+P2
			y[(depart+t)*Y+_y] = P0 * P1 + P2;
		}
	}
}

void d_intel_dot1d_mul(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	FOR(0, t, T) {
		FOR(0, _x, X) {
			float s = 0;
			FOR(0, k, Y) {
				float __x = p[k*(X+1)+_x];
				float __p = locd[(depart+t)*Y+k] * dy[(depart+t)*Y+k];
				s += __x * __p;
			}
			dx[(depart+t)*X_vars+DEPART_x+_x]   = s;
		}
	}

	//dp = x.T @ ((y-_y)*dtanh(x@p))
	FOR(0, _y, Y) {
		float dbiais = 0;
		FOR(0, _x, X) {
			float s = 0;
			FOR(0, t, T) {
				float __x = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
				float __p = x[(depart+t)*X_vars+DEPART_x+_x];
				s += __x * __p;
				if (_x == 0) {	//	Biais
					dbiais += __x;
				}
			}
			dp[_y*(X+1)+_x] = s;
		}
		dp[_y*(X+1)+(X+1-1)] = dbiais;
	}
};

//	=========================================================

void f_dot1d_mul(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	ERR("Pas ajouté le depart_plus_t")
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		intel_dot1d_mul(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst]);
	} else if (mode == 3) {
		nvidia_dot1d_mul_shared_2_16(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_dot1d_mul(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	ERR("Pas ajouté le depart_plus_t")
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		d_intel_dot1d_mul(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst],
			mdl->dy[inst],
			mdl->dy[inst-1],
			mdl->dp[inst]);
	} else if (mode == 3) {
		d_nvidia_dot1d_mul_shared_2_16(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}