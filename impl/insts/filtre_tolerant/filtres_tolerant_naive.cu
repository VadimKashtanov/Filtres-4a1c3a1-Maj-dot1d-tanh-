#include "hip/hip_runtime.h"
#include "filtres_prixs_tolerant.cuh"

#define BLOQUE_T  4//8
#define BLOQUE_B  4//8
#define BLOQUE_FB 8//16//32//8

#include "../../../impl_tmpl/tmpl_etc.cu"

static __device__ float filtre_device(float * x, float * dif_x, float * f, float * locd) {
	float s = 0;
	float f_nouveau = f[0];
	s += powf(1 + fabs(x[0] - f_nouveau), 1.0/*(0.5+0/N*0.5)*/);
	FOR(1, i, N) {
		f_nouveau = f[i];
		float Ps = 1.0;//(0.5+i/N*0.5);
		//s += powf(1 + fabs(   x[i]  -       f_nouveau    ), 0.5);
		s += powf(1 + fabs(   x[i]  -       f_nouveau    ), Ps);
	};

	s = s/(float)N-1;
	
	float y = expf(-s*s);

	locd[0] = -2*2*s*y;

	return 2*y-1;
	//return 2*filtres_f_info(y)-1;
};

static __global__ void kerd_filtre_naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		y[(0+_t)*(bloques*f_par_bloque) + _b*f_par_bloque + _f] = filtre_device(
			x     + _b*PRIXS*N_FLTR + /*(depart+_t)*/cuda_depart_plus_t*N_FLTR,
			dif_x + _b*PRIXS*N_FLTR + /*(depart+_t)*/cuda_depart_plus_t*N_FLTR,
			f     + _b*f_par_bloque*N     + _f*N,
			locd  + (0+_t)*bloques*f_par_bloque*1 + _b*f_par_bloque*1 + _f*1
		);
	}
};

void nvidia_filtres_prixs_tolerant___naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	kerd_filtre_naive<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		depart, T,
		bloques, f_par_bloque,
		x, dif_x,
		f,
		y,
		locd);
	ATTENDRE_CUDA();
}

__device__ static void d_nvidia_filtre(float * x, float * dif_x, float * f, float * locd, float * dy, float * df) {
	/*float ds = locd[0] * dy[0] / 8;
	float dd = locd[1] * dy[0] / 7;
	//
	FOR(1, i, N)
	{
		//s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		atomicAdd(&df[i], ds * 1 / (2*sqrtf(1 + fabs(x[i] - f[i]))) * (-1) * cuda_signe(x[i] - f[i]));
		//d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
		atomicAdd(&df[ i ], dd * 2 * (1 + fabs(dif_x[i] - (f[i]-f[i-1]))) * cuda_signe(dif_x[i] - (f[i]-f[i-1])) * (-1));
		atomicAdd(&df[i-1], dd * 2 * (1 + fabs(dif_x[i] - (f[i]-f[i-1]))) * cuda_signe(dif_x[i] - (f[i]-f[i-1])) * (+1));
	}
	atomicAdd(&df[0], ds * 1 / (2*sqrtf(1 + fabs(x[0] - f[0]))) * (-1) * cuda_signe(x[0] - f[0]));*/

	float ds = locd[0] * dy[0] / 8;
	//
	FOR(1, i, N)
	{
		float Ps = 1.0;//(0.5+i/N*0.5);
		
		//s += powf(1 + fabs(   x[i]  -       f_nouveau    ), (0.5+i/N*0.5));
		atomicAdd(&df[i], ds * Ps * powf(1 + fabs(x[i] - f[i]), Ps-1) * (-1) * cuda_signe(x[i] - f[i]));
	}
	float Ps = 1.0;//(0.5+0/N*0.5);
	//df[0] += ds * 1 / (2*sqrtf(1 + fabs(x[0] - f[0]))) * (-1) * signe(x[0] - f[0]);
	atomicAdd(&df[0], ds * Ps * powf(1 + fabs(x[0] - f[0]), Ps-1) * (-1) * cuda_signe(x[0] - f[0]));

};

__global__ static void  d_nvidia_kerd_filtre_naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		d_nvidia_filtre(
				x + _b*PRIXS*N_FLTR + /*(depart+_t)*/cuda_depart_plus_t*N_FLTR,
			dif_x + _b*PRIXS*N_FLTR + /*(depart+_t)*/cuda_depart_plus_t*N_FLTR,
			f     + _b*f_par_bloque*N     + _f*N,
			locd  + (0+_t)*(bloques*f_par_bloque*1) + _b*(f_par_bloque*1) + _f*1,
			dy    + (0+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			df    + _b*f_par_bloque*N     + _f*N
		);
	}
};

void d_nvidia_filtres_prixs_tolerant___naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	d_nvidia_kerd_filtre_naive<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		depart, T,
		bloques, f_par_bloque,
		x, dif_x,
		f,
		y,
		locd,
		dy,
		df);
	ATTENDRE_CUDA();
}