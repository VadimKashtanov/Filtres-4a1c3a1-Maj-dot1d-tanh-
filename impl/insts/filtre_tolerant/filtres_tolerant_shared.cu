#include "hip/hip_runtime.h"
#include "filtres_prixs_tolerant.cuh"

#define BLOQUE_T  16

#define _repete_T 16

#include "../../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_filtre_shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	uint depart_bloque_t = blockIdx.y * BLOQUE_T * _repete_T;
	uint depart_thread_t = depart_bloque_t + threadIdx.y * _repete_T;

	uint _b = blockIdx.x;
	uint _f = blockIdx.z;	//(ligne dans bloque)

	uint LIGNE  = _b;
	uint BLOQUE = _b; 

	uint thx = threadIdx.x;
	uint thy__t = threadIdx.y;

	//if (_t < T)
	__shared__ float __f__[N];
	//
	if (thy__t==0) __f__[thx]  = f[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx];
	__syncthreads();
	//
	float fi;
	fi = __f__[thx];
	//
	__shared__ float __ret[BLOQUE_T][1];	//s, d
	__shared__ float __y  [BLOQUE_T];
	//
	float xi;
	//
	uint _t;
	FOR(0, plus_t, _repete_T) {
		//
		_t = depart_thread_t + plus_t;

		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		//
		if (thx < 1) {
			__ret[thy__t][thx] = 0;
		}
		__syncthreads();
		//
		xi = x[LIGNE*PRIXS*N_FLTR + /*(depart+_t)*/cuda_depart_plus_t*N_FLTR + thx];
		//
		float Ps = 1.0;//(0.5+thx/N*0.5);
		atomicAdd(&__ret[thy__t][0], powf(1 + fabs(xi - fi), Ps));
		__syncthreads();
		//
		if (thx < 1) {
			__ret[thy__t][thx] = __ret[thy__t][thx]/(float)(8-thx) - 1.0;
		}
		__syncthreads();
		//
		if (thx < 1) {
			__y[thy__t] = expf(-__ret[thy__t][0]*__ret[thy__t][0]);
		}
		__syncthreads();
		//
		if (thx < 1) {
			locd[(0+_t)*BLOQUES*(F_PAR_BLOQUES*1) + BLOQUE*(F_PAR_BLOQUES*1) + _f*1 + thx] = -2*2*__ret[thy__t][thx]*__y[thy__t];
		}
		__syncthreads();
		//
		if (thx < 1) {
			y[(0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f] = 2*__y[thy__t] - 1;
		}
	};
};

void nvidia_filtres_prixs_tolerant___shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	ASSERT(BLOQUE_T*_repete_T <= T);
	kerd_filtre_shared<<<dim3(bloques, KERD((DIV(T,_repete_T)), BLOQUE_T), F_PAR_BLOQUES), dim3(N, BLOQUE_T,1)>>>(
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		depart, T,
		bloques,
		x, dif_x,
		f,
		y,
		locd);
	ATTENDRE_CUDA();
};

static __global__ void d_kerd_filtre_shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	uint depart_bloque_t = blockIdx.y * BLOQUE_T * _repete_T;
	uint depart_thread_t = depart_bloque_t + threadIdx.y * _repete_T;

	uint _b = blockIdx.x;
	uint _f = blockIdx.z;	//(ligne dans bloque)

	uint LIGNE  = _b;
	uint BLOQUE = _b; 

	uint thx = threadIdx.x;
	uint thy__t = threadIdx.y;

	//if (_t < T)
	__shared__ float __f__[N];
	__shared__ float __df__[N];
	//
	if (thy__t==0) {
		__f__[thx]  = f[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx];
		__df__[thx] = 0;
	}
	__syncthreads();
	//
	float fi;
	fi = __f__[thx];
	//
	__shared__ float __locd[BLOQUE_T][1];	//ds, dd
	__shared__ float __dy0[BLOQUE_T];
	//
	float xi;
	//
	uint _t;
	FOR(0, plus_t, _repete_T) {
		_t = depart_thread_t + plus_t;
		uint cuda_depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, _t);
		//
		if (thx < 1) {
			__dy0[thy__t] = dy[(0+_t)*BLOQUES*F_PAR_BLOQUES + BLOQUE*F_PAR_BLOQUES + _f];
		}
		__syncthreads();
		//
		if (thx < 1) {
			__locd[thy__t][thx] = locd[(0+_t)*BLOQUES*(F_PAR_BLOQUES*1) + BLOQUE*(F_PAR_BLOQUES*1) + _f*1 + thx] * __dy0[thy__t]/ (float)(8 - thx);
		}
		__syncthreads();
		//
		xi = x[LIGNE*PRIXS*N_FLTR + /*(depart+_t)*/cuda_depart_plus_t*N_FLTR + thx];
		//
		//atomicAdd(&__ret[thy__t][0], sqrtf(1 + fabs(xi - fi)));
		float Ps = 1.0;//(0.5+thx/N*0.5);
		atomicAdd(&__df__[thx], __locd[thy__t][0] * (Ps) * powf(1 + fabs(xi - fi), Ps-1) * (-1) * cuda_signe(xi - fi));
		__syncthreads();
	};
	__syncthreads();
	if (thy__t == 0) {
		atomicAdd(&df[BLOQUE*F_PAR_BLOQUES*N + _f*N + thx], __df__[thx]);
	}
};

void d_nvidia_filtres_prixs_tolerant___shared(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	ASSERT(BLOQUE_T*_repete_T <= T);
	d_kerd_filtre_shared<<<dim3(bloques, KERD((DIV(T,_repete_T)), BLOQUE_T), F_PAR_BLOQUES), dim3(N, BLOQUE_T,1)>>>(
		_t_MODE, GRAINE,
		X_vars, Y_vars,
		depart, T,
		bloques,
		x, dif_x,
		f,
		y,
		locd,
		dy,
		df);
	ATTENDRE_CUDA();
}