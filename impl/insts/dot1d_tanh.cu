#include "hip/hip_runtime.h"
#include "dot1d_tanh.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_dot1d_tanh(Mdl_t * mdl, uint c)
{
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	mdl->inst_POIDS        [c] = (mdl->Y[c-1]+1)*mdl->Y[c];
	mdl->inst_VARS         [c] = mdl->Y[c];
	mdl->inst_LOCDS        [c] = mdl->Y[c];
	mdl->inst_SORTIES      [c] = mdl->Y[c];
	mdl->inst_DEPART_SORTIE[c] = mdl->Y[c] - mdl->Y[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);

	FOR(0, y, Y) {
		FOR(0, x, X+1) {
			mdl->p[c][y*(X+1)+x] = (2*rnd()-1) * sqrtf(/*10.0*/ 15.0 / (X+Y));
		}
	}
};

void plume_dot1d_tanh(Mdl_t * mdl, uint c)
{
	printf("POIDS dot1d_tanh: \n");
	uint X=mdl->Y[c-1], Y=mdl->Y[c];
	FOR(0, y, Y) {
		printf("y=%i : ", y);
		FOR(0, x, X) {
			printf("%+f,", mdl->p[c][y*(X+1)+x]);
		}
		printf(" biais=%+f\n", mdl->p[c][y*(X+1)+X+1-1]);
	}
};

void intel_dot1d_tanh(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
/*#pragma omp parallel
#pragma omp for*/
	FOR(0, t, T) {
		FOR(0, _y, Y) {
			float s = p[_y*(X+1)+(X+1-1)];
			FOR(0, k, X) {
				float __x = x[(0+t)*X_vars+DEPART_x+k];
				float __p = p[_y*(X+1)+k];
				s += __x * __p;
			}
			float a = ACTIV(ACTIVATION, s);
			y[(0+t)*Y+_y]    = a;
			locd[(0+t)*Y+_y] = dACTIV(ACTIVATION, s, a);
		}
	}
}

void d_intel_dot1d_tanh(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
//#pragma omp parallel
//#pragma omp for
/*	FOR(0, t, T) {
		FOR(0, _y, Y) {
			float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
			dp[_y*(X+1)+(X+1-1)] += _locd;
			FOR(0, k, X) {
				//s += x[t*X+k] * p[y*(X+1)+k];
				dx[(depart+t)*X+k]      += _locd * p[_y*(X+1)+k];
				dp[_y*(X+1)+k] += _locd * x[(depart+t)*X+k];
			}
		}
	}*/

	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
/*#pragma omp parallel
#pragma omp for*/
	FOR(0, t, T) {
		FOR(0, _x, X) {
			//float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
			float s = 0;
			FOR(0, k, Y) {
				float __x = p[k*(X+1)+_x];//x[(depart+t)*X+k];
				float __p = locd[(depart+t)*Y+k] * dy[(depart+t)*Y+k];//p[_y*(X+1)+k];
				s += __x * __p;
			}
			dx[(depart+t)*X_vars+DEPART_x+_x]   = s;
		}
	}

	//dp = x.T @ ((y-_y)*dtanh(x@p))
/*#pragma omp parallel
#pragma omp for*/
	FOR(0, _y, Y) {
		float dbiais = 0;
		FOR(0, _x, X) {
			float s = 0;
			FOR(0, t, T) {
				float __x = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];//x[(depart+t)*X+k];
				float __p = x[(depart+t)*X_vars+DEPART_x+_x];//p[_y*(X+1)+k];
				s += __x * __p;
				if (_x == 0) {	//	Biais
					dbiais += __x;
				}
			}
			dp[_y*(X+1)+_x] = s;
		}
		dp[_y*(X+1)+(X+1-1)] = dbiais;
	}
}

//	=========================================================

void f_dot1d_tanh(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		intel_dot1d_tanh(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst]);
	} else if (mode == 1) {
		nvidia_dot1d_tanh_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 2) {
		nvidia_dot1d_tanh_shared(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 3) {
		nvidia_dot1d_tanh_shared_2_16(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void df_dot1d_tanh(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	uint X=mdl->Y[inst-1], Y=mdl->Y[inst];
	uint X_vars=mdl->inst_VARS[inst-1], Y_vars=mdl->inst_VARS[inst];
	uint depart = 0;//t0;
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	uint DEPART_x = mdl->inst_DEPART_SORTIE[inst-1];
	if (mode == 0) {
		d_intel_dot1d_tanh(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y[inst-1], mdl->y[inst],
			mdl->p[inst],
			mdl->l[inst],
			mdl->dy[inst],
			mdl->dy[inst-1],
			mdl->dp[inst]);
	} else if (mode == 1) {
		d_nvidia_dot1d_tanh_naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == 2) {
		d_nvidia_dot1d_tanh_shared(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else if (mode == 3) {
		d_nvidia_dot1d_tanh_shared_2_16(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			mdl->y__d[inst-1], mdl->y__d[inst],
			mdl->p__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dy__d[inst-1],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour cuda df(x)", mode);
	}
}