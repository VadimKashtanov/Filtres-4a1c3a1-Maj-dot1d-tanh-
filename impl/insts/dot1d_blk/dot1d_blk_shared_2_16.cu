#include "hip/hip_runtime.h"
#include "dot1d_blk.cuh"

/*	Difference :
	Au lieux de directement deriver avec que des atomicAdd le
__shared__ noyau, on fait la méthode que j'avais avant
ou on fait une autre opération pour calc dx et dp.

	Mathématiquement ca correspond a deriver y=X@P+B
en dX=p@dY.T
	dx = (p @ ((y-_y)*dtanh(x@p)).T).T
	dp = x.T @ ((y-_y)*dtanh(x@p))
*/

#define BLOQUE 16
#define BLOQUE_MAX 16

static __global__ void kerd_blk_stricte_16__shared2(
	uint X_blk, uint Y_blk, uint P_blk,
	uint depart_y, uint depart_x, uint depart_p,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	// <KERD(T, BLOQUE), KERD(Y,BLOQUE)>
	// <         BLOQUE,         BLOQUE>

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _y = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, X_blk/BLOQUE) {
		__partage__x[thy][thx] = x[depart_x+(depart+_t)*( X_vars ) + DEPART_x +d*BLOQUE + thx];
		__partage__p[thy][thx] = p[depart_p+_y*(X_blk+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

#define __partage__b __partage__x[0]

	if (thy == 0) __partage__b[thx] = p[depart_p+_y*(X_blk+1) + (X_blk+1-1)];
	__syncthreads();

	s = (s + __partage__b[thx]);
	float a = dot1d_blk_ACTIV(dot1d_blk_ACTIVATION, s);
	   y[(0+_t)*Y + depart_y+_y] = a;
	locd[(0+_t)*Y + depart_y+_y] = dot1d_blk_dACTIV(dot1d_blk_ACTIVATION, s,a);
};

void nvidia_dot1d_blk_shared_2_16(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	//
	ASSERT(X % (DOT1D_BLK_BLOQUES*BLOQUE_MAX) == 0);
	ASSERT(Y % (DOT1D_BLK_BLOQUES*BLOQUE)     == 0);
	//
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	if (T%BLOQUE!=0) ERR("ATTENTION T%%16 != 0 (T=%i)", T);
	if (X_blk%BLOQUE==0 && Y_blk%BLOQUE==0 && T%BLOQUE==0) {
		FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
			uint depart_y = dot1d_blk * Y_blk;	//depat d'un bloque en x
			uint depart_x = dot1d_blk * X_blk;	// 					en y
			uint depart_p = dot1d_blk * P_blk;
			kerd_blk_stricte_16__shared2<<<dim3(KERD(Y_blk, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
				X_blk, Y_blk, P_blk,
				depart_y, depart_x, depart_p,
				X_vars, Y_vars,
				X, Y,
				depart, T,
				DEPART_x,
				x, y,
				p,
				locd);
		}
		ATTENDRE_CUDA();
	} else {
		nvidia_dot1d_blk_naive(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd);
	}
}

static __global__ void kerd_blk_stricte_16__shared2____dx(
	uint X_blk, uint Y_blk, uint P_blk,
	uint depart_y, uint depart_x, uint depart_p,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T

	__shared__ float __partage__x[BLOQUE][BLOQUE];
	__shared__ float __partage__p[BLOQUE][BLOQUE];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	//uint _y = thx + blockIdx.x * blockDim.x;
	uint _x = thx + blockIdx.x * blockDim.x;
	uint _t = thy + blockIdx.y * blockDim.y;

	float s = 0;

	FOR(0, d, Y_blk/BLOQUE) {
		__partage__x[thy][thx] = locd[depart_y+(depart+_t)*Y+d*BLOQUE+thx] * dy[depart_y+(depart+_t)*Y+d*BLOQUE+thx];//x[(depart+_t)*( X ) + d*BLOQUE + thx];
		__partage__p[thy][thx] = p[depart_p+(d*BLOQUE+thy)*(X_blk+1) + _x];//p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	//#pragma unroll
		FOR(0, i, BLOQUE) s += __partage__x[thy][i] * __partage__p[i][thx];
		__syncthreads();
	};

	dx[(depart+_t)*X_vars+DEPART_x +depart_x+_x] = s;
	//printf("s=%f\n", s);
};


static __global__ void kerd_blk_stricte_32__shared2____dp(
	uint X_blk, uint Y_blk, uint P_blk,
	uint depart_y, uint depart_x, uint depart_p,
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	//dp = x.T @ ((y-_y)*dtanh(x@p))

	__shared__ float __partage__x[BLOQUE_MAX][BLOQUE_MAX];
	__shared__ float __partage__p[BLOQUE_MAX][BLOQUE_MAX];

	uint thx = threadIdx.x;
	uint thy = threadIdx.y;

	uint _x = thx + blockIdx.x * blockDim.x;
	uint _y = thy + blockIdx.y * blockDim.y;

	float s = 0;
	float biais = 0;

	uint d = blockIdx.z;
	//FOR(0, d, T/BLOQUE) {
		__partage__x[thy][thx] = locd[depart_y+(depart+d*BLOQUE_MAX+thx)*Y+_y] * dy[depart_y+(depart+d*BLOQUE_MAX+thx)*Y+_y];//x[(depart+_t)*( X ) + d*BLOQUE + thx];
		__partage__p[thy][thx] = x[depart_x+(depart+(d*BLOQUE_MAX+thy))*X_vars+DEPART_x +_x];//p[_y*(X+1) + d*BLOQUE + thy];
		__syncthreads();

	#pragma unroll
		FOR(0, i, BLOQUE_MAX) {
			s += __partage__x[thy][i] * __partage__p[i][thx];
			if (_x == 0) biais += __partage__x[thy][i];
		}
		__syncthreads();
	//};

#define __partage__b __partage__x[0]

	//if (thy == 0) __partage__b[thx] = p[_y*(X+1) + (X+1-1)];
	if (_x == 0) atomicAdd(&dp[depart_p+_y*(X_blk+1) + (X_blk+1-1)], biais);
	__syncthreads();

	atomicAdd(&dp[depart_p+_y*(X_blk+1)+_x], s);
};

void d_nvidia_dot1d_blk_shared_2_16(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	ASSERT(X % (DOT1D_BLK_BLOQUES*BLOQUE_MAX) == 0);
	ASSERT(Y % (DOT1D_BLK_BLOQUES*BLOQUE)     == 0);
	//
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	if (T%(MAX2(BLOQUE_MAX,BLOQUE))!=0) ERR("ATTENTION T%%%i != 0 (T=%i)", T, (MAX2(BLOQUE_MAX,BLOQUE)));
	if (X_blk%BLOQUE==0 && Y_blk%BLOQUE==0 && T%BLOQUE==0) {
		FOR(0, dot1d_blk, DOT1D_BLK_BLOQUES) {
			uint depart_y = dot1d_blk * Y_blk;	//depat d'un bloque en x
			uint depart_x = dot1d_blk * X_blk;	// 					en y
			uint depart_p = dot1d_blk * P_blk;
			//
			kerd_blk_stricte_16__shared2____dx<<<dim3(KERD(X_blk, BLOQUE), KERD(T, BLOQUE)), dim3(BLOQUE, BLOQUE)>>>(
				X_blk, Y_blk, P_blk,
				depart_y, depart_x, depart_p,
				X_vars, Y_vars,
				X, Y,
				depart, T,
				DEPART_x,
				x, y,
				p,
				locd,
				dy,
				dx,
				dp);
			kerd_blk_stricte_32__shared2____dp<<<dim3(KERD(X_blk, BLOQUE_MAX), KERD(Y_blk, BLOQUE_MAX), DIV(T,BLOQUE_MAX)), dim3(BLOQUE_MAX, BLOQUE_MAX,1)>>>(
				X_blk, Y_blk, P_blk,
				depart_y, depart_x, depart_p,
				X_vars, Y_vars,
				X, Y,
				depart, T,
				DEPART_x,
				x, y,
				p,
				locd,
				dy,
				dx,
				dp);
		}
		ATTENDRE_CUDA();
	} else {
		d_nvidia_dot1d_blk_naive(
			X_vars, Y_vars,
			X, Y,
			depart, T,
			DEPART_x,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	}
}