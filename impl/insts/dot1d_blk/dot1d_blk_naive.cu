#include "hip/hip_runtime.h"
#include "dot1d_blk.cuh"

#define BLOQUE_T 32
#define BLOQUE_Y 32

static __global__ void kerd_dot1d_blk_naive(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;
	//
	if (_t < T && _y < Y) {
		uint dot1d_blk = (_y-_y%Y_blk)/Y_blk;
		//
		uint depart_y = dot1d_blk * Y_blk;
		uint depart_x = dot1d_blk * X_blk;
		uint depart_p = dot1d_blk * P_blk;
		//
		float s = p[depart_p+_y*(X_blk+1) + (X_blk+1-1)];
		FOR(0, i, X_blk) s += x[depart_x+(depart+_t)*X_vars + DEPART_x + i] * p[depart_p+_y*(X_blk+1) + i];
		float a = dot1d_blk_ACTIV(dot1d_blk_ACTIVATION, s);
		y[(depart+_t)*Y + depart_y+_y] = a;
		locd[(depart+_t)*Y + depart_y+_y] = dot1d_blk_dACTIV(dot1d_blk_ACTIVATION, s,a);
	}
};

void nvidia_dot1d_blk_naive(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd)
{
	kerd_dot1d_blk_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
}

//	============================= Derivation ==============================

static __global__ void kerd_deriv_dot1d_blk_naive(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;
	//
	uint X_blk = X / DOT1D_BLK_BLOQUES;
	uint Y_blk = Y / DOT1D_BLK_BLOQUES;
	uint P_blk = (X_blk+1)*Y_blk;

	if (_t < T && _y < Y) {
		uint dot1d_blk = (_y-_y%Y_blk)/Y_blk;
		//
		uint depart_y = dot1d_blk * Y_blk;
		uint depart_x = dot1d_blk * X_blk;
		uint depart_p = dot1d_blk * P_blk;
		//
		float _locd = locd[(depart+_t)*Y + depart_y+_y] * dy[(depart+_t)*Y + depart_y+_y];
		atomicAdd(&dp[depart_p+_y*(X_blk+1) + (X_blk+1-1)], _locd);
		FOR(0, i, X_blk) {
			atomicAdd(&dx[(depart+_t)*X_vars + DEPART_x +depart_x+ i], _locd * p[depart_p+_y*(X_blk+1) + i]);
			atomicAdd(&dp[depart_p+_y*(X_blk+1) + i], _locd * x[(depart+_t)*X_vars + DEPART_x +depart_x+ i]);
		}
	}
};

void d_nvidia_dot1d_blk_naive(
	uint X_vars, uint Y_vars,
	uint X, uint Y,
	uint depart, uint T,
	uint DEPART_x,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	kerd_deriv_dot1d_blk_naive<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_Y)>>>(
		X_vars, Y_vars,
		X, Y,
		depart, T,
		DEPART_x,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};