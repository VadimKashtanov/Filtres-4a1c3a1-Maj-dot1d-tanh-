#include "hip/hip_runtime.h"
#include "filtres_prixs_tolerant.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void cree_filtres_prixs_tolerant(Mdl_t * mdl, uint c)
{
	mdl->inst_POIDS        [c] = BLOQUES*F_PAR_BLOQUES*N;
	mdl->inst_VARS         [c] = mdl->Y[c];
	mdl->inst_LOCDS        [c] = 1*mdl->Y[c];
	mdl->inst_SORTIES      [c] = mdl->Y[c];
	mdl->inst_DEPART_SORTIE[c] = mdl->Y[c] - mdl->Y[c];
	//
	mdl->p[c] = alloc<float>(mdl->inst_POIDS[c]);
	FOR(0, i, mdl->inst_POIDS[c])
		mdl->p[c][i] = (2*rnd()-1) * 1.0;
};

void plume_filtres_prixs_tolerant(Mdl_t * mdl, uint c)
{
	printf("POIDS FILTRES: \n");
	FOR(0, b, BLOQUES) {
		FOR(0, f, F_PAR_BLOQUES) {
			printf("bloque=%i f=%i :", b, f);
			FOR(0, i, N)
				printf("%+f, ", mdl->p[c][b*F_PAR_BLOQUES*N + f*N + i]);
			printf("\n");
		}
	}
};

static float filtre(float * x, float * dif_x, float * f, float * locd) {
	float s = 0;
	float f_nouveau = f[0];
	s += powf(1 + fabs(x[0] - f_nouveau), 1.0/*(0.5+0/N*0.5)*/);
	FOR(1, i, N) {
		f_nouveau = f[i];
		float Ps = 1.0;//(0.5+i/N*0.5);
		//s += powf(1 + fabs(   x[i]  -       f_nouveau    ), 0.5);
		s += powf(1 + fabs(   x[i]  -       f_nouveau    ), Ps);
	};

	s = s/(float)N-1;
	
	float y = expf(-s*s);

	locd[0] = -2*2*s*y;

	return 2*y-1;
	//return 2*filtres_f_info(y)-1;
};

void intel_filtres_prixs_tolerant___naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd)
{
	FOR(0, t, T) {
		uint depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);
		FOR(0, b, bloques) {
			FOR(0, _f, f_par_bloque) {
				y[(0+t)*bloques*f_par_bloque + b*f_par_bloque + _f] = filtre(
						x + b*PRIXS*N_FLTR + /*(depart+t)*/depart_plus_t*N_FLTR,
					dif_x + b*PRIXS*N_FLTR + /*(depart+t)*/depart_plus_t*N_FLTR,
					f     + b*f_par_bloque*N     + _f*N,
					locd  + (0+t)*(bloques*f_par_bloque*1) + b*(f_par_bloque*1) + _f*1
				);
			}
		}
	}
};

static void d_filtre(float * x, float * dif_x, float * f, float * locd, float * dy, float * df) {
	float ds = locd[0] * dy[0] / 8;
	//
	FOR(1, i, N)
	{
		float Ps = 1.0;//(0.5+i/N*0.5);
		/*//s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		df[i] += ds * 1 / (2*sqrtf(1 + fabs(x[i] - f[i]))) * (-1) * signe(x[i] - f[i]);
		//d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
		df[ i ] += dd * 2 * (1 + fabs(dif_x[i] - (f[i]-f[i-1]))) * signe(dif_x[i] - (f[i]-f[i-1])) * (-1);
		df[i-1] += dd * 2 * (1 + fabs(dif_x[i] - (f[i]-f[i-1]))) * signe(dif_x[i] - (f[i]-f[i-1])) * (+1);*/

		//s += powf(1 + fabs(   x[i]  -       f_nouveau    ), (0.5+i/N*0.5));
		df[i] += ds * Ps * powf(1 + fabs(x[i] - f[i]), Ps-1) * (-1) * signe(x[i] - f[i]);
	}
	float Ps = 1.0;//(0.5+0/N*0.5);
	//df[0] += ds * 1 / (2*sqrtf(1 + fabs(x[0] - f[0]))) * (-1) * signe(x[0] - f[0]);
	df[0] += ds * Ps * powf(1 + fabs(x[0] - f[0]), Ps-1) * (-1) * signe(x[0] - f[0]);
};

void  d_intel_filtres_prixs_tolerant___naive(
	uint _t_MODE, uint GRAINE,
	uint X_vars, uint Y_vars,
	uint depart, uint T,
	uint bloques, uint f_par_bloque,
	float * x, float * dif_x,
	float * f,
	float * y,
	float * locd,
	float * dy,
	float * df)
{
	FOR(0, t, T) {
		uint depart_plus_t = t_MODE_GENERALE(_t_MODE, GRAINE, depart, DEPART, FIN, t);
		FOR(0, b, bloques) {
			FOR(0, _f, f_par_bloque) {
				d_filtre(
						x + b*PRIXS*N_FLTR + /*(depart+t)*/depart_plus_t*N_FLTR,
					dif_x + b*PRIXS*N_FLTR + /*(depart+t)*/depart_plus_t*N_FLTR,
					f     + b*f_par_bloque*N     + _f*N,
					locd  + (     0+t)*(bloques*f_par_bloque*1) + b*(f_par_bloque*1) + _f*1,
					dy    + (     0+t)*(bloques*f_par_bloque  ) + b*(f_par_bloque  ) + _f,
					df    + b*f_par_bloque*N     + _f*N
				);
			}
		}
	}
};

void f_filtres_prixs_tolerant(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	uint depart = t0;
	uint X_vars=0, Y_vars=mdl->inst_VARS[inst];
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	if (mode == 0) {
		intel_filtres_prixs_tolerant___naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			depart, T,
			BLOQUES, F_PAR_BLOQUES,
			mdl->normalisee, mdl->dif_normalisee,
			mdl->p[inst],
			mdl->y[inst],
			mdl->l[inst]);
	} else if (mode == 1/* || mode == 2 || mode == 3*/) {
		nvidia_filtres_prixs_tolerant___naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			depart, T,
			BLOQUES, F_PAR_BLOQUES,
			mdl->normalisee__d, mdl->dif_normalisee__d,
			mdl->p__d[inst],
			mdl->y__d[inst],
			mdl->l__d[inst]);
	} else if (mode == 2 || mode == 3) {
		nvidia_filtres_prixs_tolerant___shared(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			depart, T,
			BLOQUES, F_PAR_BLOQUES,
			mdl->normalisee__d, mdl->dif_normalisee__d,
			mdl->p__d[inst],
			mdl->y__d[inst],
			mdl->l__d[inst]);
	} else {
		ERR("Pas de mode %i pour mes filtres", mode);
	};
};

void df_filtres_prixs_tolerant(Mdl_t * mdl, uint inst, uint mode, uint t0, uint t1, uint _t_MODE, uint GRAINE) {
	uint depart = t0;
	uint X_vars=0, Y_vars=mdl->inst_VARS[inst];
	uint T = (t1-t0);
	ASSERT(T == mdl->T);
	if (mode == 0) {
		d_intel_filtres_prixs_tolerant___naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			depart, T,
			BLOQUES, F_PAR_BLOQUES,
			mdl->normalisee, mdl->dif_normalisee,
			mdl->p[inst],
			mdl->y[inst],
			mdl->l[inst],
			mdl->dy[inst],
			mdl->dp[inst]);
	} else if (mode == 1/* || mode == 2 || mode == 3*/) {
		d_nvidia_filtres_prixs_tolerant___naive(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			depart, T,
			BLOQUES, F_PAR_BLOQUES,
			mdl->normalisee__d, mdl->dif_normalisee__d,
			mdl->p__d[inst],
			mdl->y__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dp__d[inst]);
	} else if (mode == 2 || mode == 3) {
		d_nvidia_filtres_prixs_tolerant___shared(
			_t_MODE, GRAINE,
			X_vars, Y_vars,
			depart, T,
			BLOQUES, F_PAR_BLOQUES,
			mdl->normalisee__d, mdl->dif_normalisee__d,
			mdl->p__d[inst],
			mdl->y__d[inst],
			mdl->l__d[inst],
			mdl->dy__d[inst],
			mdl->dp__d[inst]);
	} else {
		ERR("Pas de mode %i pour mes filtres", mode);
	}
};