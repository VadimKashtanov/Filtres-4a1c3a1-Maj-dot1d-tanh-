#include "hip/hip_runtime.h"
#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

#include "marchee.cuh"

static float filtre(uint inst0, uint depart, float * x, float * f, uint intervalle, uint type_norme, float _min, float _max) {
	float normer_x[N];
	//
	FOR(0, i, N) normer_x[i] = x[depart - (i)*intervalle];
	//
	if (type_norme == NORME_CLASSIQUE) {
		_min=normer_x[0];
		_max=normer_x[0];
		//
		FOR(1, i, N) {
			float a = normer_x[i];
			if (a > _max) _max = a;
			if (a < _min) _min = a;
		}
	} else if (type_norme == NORME_THEORIQUE) {
		// rien
	} else if (type_norme == NORME_RELATIVE) {
		_max=fabs(normer_x[0]);
		//
		FOR(1, i, N) {
			float a = fabs(normer_x[i]);
			if (a > _max) _max = a;
		}
		_max = +_max;
		_min = -_max;
	} else {
		ERR("type_norme == %i", type_norme);
	}
	//
	FOR(0, i, N) normer_x[i] = (normer_x[i]-_min)/(_max-_min);
	if (inst0 == 0) {
		//
		float s = 0, d = 0;
		float f_nouveau = f[0];
		float x_nouveau = normer_x[0];
		//
		float Ps = (0.5+0/N*0.5);
		s += powf(1 + fabs(x_nouveau - f_nouveau), Ps);
		//
		float f_avant = f_nouveau;
		float x_avant = x_nouveau;
		FOR(1, i, N) {
			f_nouveau = f[i];
			x_nouveau = normer_x[i];
			//
			float Ps = (0.5+i/N*0.5);
			float Pd = (1.0+i/N*1.0);
			//
			s += powf(1 + fabs(  x_nouveau   -   f_nouveau  ), Ps);
			d += powf(1 + fabs((x_nouveau-x_avant) - (f_nouveau-f_avant)), Pd);
			f_avant   = f_nouveau;
			x_avant   = x_nouveau;
		};

		s = s/(float)N-1;
		d = d/(float)(N-1)-1;

		return 2*expf(-s*s-d*d)-1;
	} else if (inst0 == 1) {
		//
		float s = 0;
		float f_nouveau = f[0];
		float x_nouveau = normer_x[0];
		//
		float Ps = 1.0;//(0.5+0/N*0.5);
		s += powf(1 + fabs(x_nouveau - f_nouveau), Ps);
		//
		float x_avant = x_nouveau;
		FOR(1, i, N) {
			f_nouveau = f[i];
			x_nouveau = normer_x[i];
			//
			float Ps = 1.0;//(0.5+i/N*0.5);
			//
			s += powf(1 + fabs(  x_nouveau   -   f_nouveau  ), Ps);
			x_avant   = x_nouveau;
		};

		s = s/(float)N-1;

		return 2*expf(-s*s)-1;
	} else {
		ERR("Pas possible")
	}
	return 0;
};

/*static __global__ void simple_DOT1D(
	float * y_nouveau, float * y_avant, float * poids,
	uint X, uint Y, uint PRIXS_bitget, uint T)
{
	uint y = threadIdx.x + blockIdx.x + blockDim.x;
	uint t = DEPART + threadIdx.y + blockIdx.y + blockDim.y;
	if (y < Y && t < PRIXS_bitget) {
		float s = poids[(X+1)*y + X-1+1];
		FOR(0, j, X) s += poids[(X+1)*y + j] * y_avant[t*MAX_Y + j];
		y_nouveau[t*MAX_Y + y] = tanh(s);
	}
};*/

int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	//
	FILE * fp = fopen(argv[1], "rb");
	//
	uint Y[C];
	FREAD(Y, sizeof(uint), C, fp);
	uint insts[C];
	FREAD(insts, sizeof(uint), C, fp);
	uint cibles[C];
	FREAD(cibles, sizeof(uint), C, fp);
	uint depart_future[C];
	FREAD(depart_future, sizeof(uint), C, fp);
	//
	//
	//
	//
	uint PRIXS_bitget;
	FREAD(&PRIXS_bitget, sizeof(uint), 1, fp);
	uint intervalles[BLOQUES];
	FREAD(intervalles, sizeof(uint), BLOQUES, fp);
	//
	//
	//
	uint type_norme[BLOQUES];
	float _min[BLOQUES], _max[BLOQUES];
	FREAD(type_norme, sizeof(uint), BLOQUES, fp);
	FREAD(_min,       sizeof(float), BLOQUES, fp);
	FREAD(_max,       sizeof(float), BLOQUES, fp);
	//
	//
	//
	float * lignes = alloc<float>(PRIXS_bitget*BLOQUES);
	FREAD(lignes, sizeof(float), PRIXS_bitget*BLOQUES, fp);
	//
	float * poids[C];
	float * poids_cuda[C];
	FOR(0, c, C) {
		uint POIDS;
		FREAD(&POIDS, sizeof(uint), 1, fp);
		poids[c] = alloc<float>(POIDS);
		FREAD(poids[c], sizeof(float), POIDS, fp);
		//
		poids_cuda[c] = cpu_vers_gpu<float>(poids[c], POIDS);
	}
	//
	fclose(fp);

	//	------------- Calcule ----------------
	float * y_avant   = alloc<float>( PRIXS_bitget*MAX_Y );
	float * y_nouveau = alloc<float>( PRIXS_bitget*MAX_Y );
	//
	float * y_avant_cuda   = cudalloc<float>(PRIXS_bitget*MAX_Y);
	float * y_nouveau_cuda = cudalloc<float>(PRIXS_bitget*MAX_Y);
	//
	//#pragma omp parallel
	//#pragma omp for
	FOR(0, f, BLOQUES*F_PAR_BLOQUES) {
		uint b = (f - (f % F_PAR_BLOQUES)) / F_PAR_BLOQUES;
		FOR(DEPART, t, PRIXS_bitget) {
			y_nouveau[t*MAX_Y + f] = filtre(
				insts[0],
				//
				b*PRIXS_bitget + t,
				lignes,
				poids[0] + f*N,
				intervalles[b],
				type_norme[b],
				_min[b], _max[b]
			);
		}
	};
	FOR(0, i, PRIXS_bitget*MAX_Y) y_avant[i] = y_nouveau[i];
	//
	CONTROLE_CUDA(hipMemcpy(y_avant_cuda, y_avant, PRIXS_bitget*MAX_Y*sizeof(float), hipMemcpyHostToDevice));
	//
	FOR(1, c, C) {
		if (insts[c] == DOT1D) {
#include "dot1d.cuh"
			uint X = Y[c-1];
			//#pragma omp parallel
			//#pragma omp for
			FOR(0, i, Y[c]) {
				FOR(DEPART, t, PRIXS_bitget) {
					float s = poids[c][(X+1)*i + X-1+1];
					FOR(0, j, X) s += poids[c][(X+1)*i + j] * y_avant[t*MAX_Y + j];
					y_nouveau[t*MAX_Y + i] = ACTIV(ACTIVATION, s);//tanh(s);
				};
			};
			/*uint T = (PRIXS_bitget-DEPART);
			simple_DOT1D<<<dim3(KERD(Y[c], 16), KERD(T,16)), dim3(16,16)>>>(
				y_nouveau_cuda, y_avant_cuda, poids_cuda[c],
				X, Y[c], PRIXS_bitget, T);
			ATTENDRE_CUDA();*/
			//
		} else if (insts[c] == DOT1D_BLK) {
#include "dot1d_blk.cuh"
			ERR("a implementer");
			//
			uint  X = Y[c-1];
			uint _Y = Y[ c ];
			//
			uint X_blk =  X / DOT1D_BLK_BLOQUES;
			uint Y_blk = _Y / DOT1D_BLK_BLOQUES;
			uint P_blk =  ( X_blk + 1 ) * Y_blk;
			//
			FOR(DEPART, t, PRIXS_bitget) {
				FOR(0, blk, DOT1D_BLK_BLOQUES) {
					//
					uint depart_y = blk * Y_blk;
					uint depart_x = blk * X_blk;
					uint depart_p = blk * P_blk;
					//
					FOR(0, y, Y_blk) {
						float s = poids[c][depart_p + (X_blk+1)*y + (X_blk+1)+1];
						FOR(0, j, X_blk)
							s += poids[c][depart_p + (X_blk+1)*y] * y_avant[t*MAX_Y + depart_x + j];
						y_nouveau[t*MAX_Y + depart_y + y] = tanh(s);
					};
				};
			}
		} else {
			ERR("Inst = %i", insts[c]);
		}

		/*#pragma omp parallel
		#pragma omp for*/
		//#pragma omp parallel
		//#pragma omp for
		FOR(0, i, PRIXS_bitget*MAX_Y) y_avant[i] = y_nouveau[i];
		//
		CONTROLE_CUDA(hipMemcpy(y_avant_cuda, y_nouveau_cuda, PRIXS_bitget*MAX_Y*sizeof(float), hipMemcpyDeviceToDevice));
		//CONTROLE_CUDA(hipMemcpy(y_avant_cuda, y_avant, PRIXS_bitget*MAX_Y*sizeof(float), hipMemcpyHostToDevice));
	};

	//CONTROLE_CUDA(hipMemcpy(y_nouveau, y_nouveau_cuda, PRIXS_bitget*MAX_Y*sizeof(float), hipMemcpyDeviceToHost));

	//	---------- Ecrire Resultat ----------
	fp = fopen(argv[1], "wb");
	//
	float res[PRIXS_bitget];
	FOR(DEPART, t, PRIXS_bitget) res[t] = y_nouveau[t*MAX_Y + 0];
	FWRITE(res+DEPART, sizeof(float), (PRIXS_bitget-DEPART), fp);
	//
	fclose(fp);
}